#include "hip/hip_runtime.h"
/*
 * This file is part of the ddafa reconstruction program.
 *
 * Copyright (C) 2016 Helmholtz-Zentrum Dresden-Rossendorf
 *
 * Licensed under the EUPL, Version 1.1 or - as soon they will be approved by
 * the European Commission - subsequent version of the EUPL (the "Licence");
 * You may not use this work except in compliance with the Licence.
 * You may obtain a copy of the Licence at:
 *
 * http://ec.europa.eu/idabc/eupl
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the Licence is distributed on an "AS IS" basis,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the Licence for the specific language governing permissions and
 * limitations under the Licence.
 *
 * Date: 18 August 2016
 * Authors: Jan Stephan
 */

#include <cmath>
#include <cstddef>
#include <cstdint>
#include <functional>
#include <numeric>
#include <utility>

#include <boost/log/trivial.hpp>

#include <hipfft/hipfft.h>

#include <ddrf/cuda/algorithm.h>
#include <ddrf/cuda/coordinates.h>
#include <ddrf/cuda/launch.h>
#include <ddrf/cuda/memory.h>
#include <ddrf/cuda/sync_policy.h>
#include <ddrf/cuda/utility.h>
#include <ddrf/cufft/plan.h>

#include "exception.h"
#include "geometry.h"
#include "filter_stage.h"

namespace ddafa
{
    namespace
    {
        __global__ void check(const float* in, std::uint32_t dim_x, std::uint32_t dim_y, std::size_t pitch)
        {
            auto x = ddrf::cuda::coord_x();
            auto y = ddrf::cuda::coord_y();

            if(x < dim_x && y < dim_y)
            {
                auto row = reinterpret_cast<const float*>(reinterpret_cast<const char*>(in) + y * pitch);

                if(x == 508 && y == 200)
                    printf("value = %f\n", row[x]);
            }
        }

        __global__ void filter_creation_kernel(float* __restrict__ r, const std::int32_t* __restrict__ j, std::uint32_t size, float tau)
        {
            auto x = ddrf::cuda::coord_x();

            /*
             * r(j) with j = [ -(filter_length - 2) / 2, ..., 0, ..., filter_length / 2 ]
             * tau = horizontal pixel distance
             *
             *          1/8 * 1/(tau^2)                     for j = 0
             * r(j) = { 0                                   for even j
             *          -(1 / (2 * j^2 * pi^2 * tau^2))     for odd j
             */
            if(x < size)
            {
                if(j[x] == 0) // does j = 0?
                    r[x] = (1.f / 8.f) * (1.f / powf(tau, 2.f));
                else // j != 0
                {
                    if(j[x] % 2 == 0) // is j even?
                        r[x] = 0.f;
                    else // j is odd
                        r[x] = -(1.f / (2.f * powf(j[x], 2.f) * powf(M_PI, 2.f) * powf(tau, 2.f)));
                }
            }
        }

        auto create_filter(std::uint32_t filter_size, float tau) -> ddrf::cuda::device_ptr<float>
        {
            /*
             * for a more detailed description see filter_creation_kernel
             */
            // create j on the host and fill it with values from -(filter_size_ - 2) / 2 to filter_size / 2
            auto h_j = ddrf::cuda::make_unique_pinned_host<std::int32_t>(filter_size);
            auto size = static_cast<std::int32_t>(filter_size);
            auto j = -(size - 2) / 2;
            std::iota(h_j.get(), h_j.get() + filter_size, j);
            BOOST_LOG_TRIVIAL(debug) << "Host filter creation succeeded";

            // create j on the device and copy j from the host to the device
            auto d_j = ddrf::cuda::make_unique_device<std::int32_t>(filter_size);
            ddrf::cuda::copy(ddrf::cuda::sync, d_j, h_j, filter_size);
            BOOST_LOG_TRIVIAL(debug) << "Copied filter from host to device";

            // create r on the device
            auto d_r  = ddrf::cuda::make_unique_device<float>(filter_size);

            // calculate the filter values
            ddrf::cuda::launch(filter_size, filter_creation_kernel, d_r.get(), static_cast<const std::int32_t*>(d_j.get()), filter_size, tau);
            BOOST_LOG_TRIVIAL(debug) << "Device filter creation succeeded";

            BOOST_LOG_TRIVIAL(debug) << "Filter creation complete";
            return d_r;
        }

        __global__ void k_creation_kernel(hipfftComplex* __restrict__ data, std::uint32_t filter_size, float tau)
        {
            auto x = ddrf::cuda::coord_x();
            if(x < filter_size)
            {
                auto result = tau * fabsf(sqrtf(powf(data[x].x, 2.f) + powf(data[x].y, 2.f)));

                data[x].x = result;
                data[x].y = result;
            }
        }

        auto create_k(std::uint32_t size, float tau) -> ddrf::cuda::device_ptr<hipfftComplex>
        {
            auto r = create_filter(size, tau);

            auto size_trans = size / 2 + 1;
            auto k = ddrf::cuda::make_unique_device<hipfftComplex>(size_trans);

            auto n = static_cast<int>(size);

            auto plan = ddrf::cufft::plan<HIPFFT_R2C>{n};
            plan.execute(r.get(), k.get());

            ddrf::cuda::launch(size_trans, k_creation_kernel, k.get(), size_trans, tau);

            return k;
        }

        __global__ void filter_application_kernel(hipfftComplex* __restrict__ data, const hipfftComplex* __restrict__ filter,
                                                    std::uint32_t filter_size, std::uint32_t data_height, std::size_t pitch)
        {
            auto x = ddrf::cuda::coord_x();
            auto y = ddrf::cuda::coord_y();

            if((x < filter_size) && (y < data_height))
            {
                auto row = reinterpret_cast<hipfftComplex*>(reinterpret_cast<char*>(data) + y * pitch);

                row[x].x *= filter[x].x;
                row[x].y *= filter[x].y;
            }
        }

        auto apply_filter(hipfftComplex* in, const hipfftComplex* k, std::uint32_t x, std::uint32_t y, std::size_t pitch, hipStream_t stream) -> void
        {
            ddrf::cuda::launch_async(stream, x, y, filter_application_kernel, in, k, x, y, pitch);
        }

        __global__ void normalization_kernel(hipfftReal* dst, std::size_t dst_pitch,
                                             const hipfftReal* src, std::size_t src_pitch,
                                             std::uint32_t width, std::uint32_t height, std::uint32_t filter_size)
        {
            auto x = ddrf::cuda::coord_x();
            auto y = ddrf::cuda::coord_y();

            if((x < width) && (y < height))
            {
                auto dst_row = reinterpret_cast<hipfftReal*>(reinterpret_cast<char*>(dst) + y * dst_pitch);
                auto src_row = reinterpret_cast<const hipfftReal*>(reinterpret_cast<const char*>(src) + y * src_pitch);

                dst_row[x] = src_row[x] / filter_size;
            }
        }

        template <class In>
        auto normalize(In& in, std::uint32_t filter_size) -> void
        {
            ddrf::cuda::launch_async(in.stream, in.width, in.height,
                                        normalization_kernel,
                                        in.ptr.get(), in.ptr.pitch(),
                                        static_cast<const hipfftReal*>(in.ptr.get()), in.ptr.pitch(),
                                        in.width, in.height, filter_size);
        }

        template <class In, class Out>
        auto expand(const In& in, Out& out, std::uint32_t x, std::uint32_t y) -> void
        {
            // reset expanded projection
            ddrf::cuda::fill(ddrf::cuda::async, out, 0, in.stream, x, y);

            // copy original projection to expanded projection
            ddrf::cuda::copy(ddrf::cuda::async, out, in.ptr, in.stream, in.width, in.height);
        }

        template <class In, class Out>
        auto shrink(const In& in, Out& out) -> void
        {
            ddrf::cuda::copy(ddrf::cuda::async, out.ptr, in, out.stream, out.width, out.height);
        }

        template <class In, class Out, class Plan>
        auto transform(In* in, Out* out, Plan& plan, hipStream_t stream) -> void
        {
            plan.set_stream(stream);
            plan.execute(in, out);
        }

    }

    filter_stage::filter_stage(int device) noexcept
    : device_{device}
    {}

    auto filter_stage::assign_task(task t) noexcept -> void
    {
        filter_size_ = static_cast<std::uint32_t>(2 * std::pow(2, std::ceil(std::log2(t.det_geo.n_row))));
        n_col_ = t.det_geo.n_col;
        tau_ = t.det_geo.l_px_row;
    }

    auto filter_stage::run() -> void
    {
        auto sre = stage_runtime_error{"filter_stage::run() failed"};

        try
        {
            ddrf::cuda::set_device(device_);

            // create filter
            auto k = create_k(filter_size_, tau_);

            // dimensionality of the FFT - 1D in this case
            constexpr auto rank = 1;

            // size of the FFT for each dimension
            auto n = static_cast<int>(filter_size_);

            // we are executing a batched FFT -> set batch size
            auto batch = static_cast<int>(n_col_);

            // allocate memory for expanded projection (projection width -> filter_size_)
            auto p_exp = ddrf::cuda::make_unique_device<float>(filter_size_, n_col_);

            // allocate memory for transformed projection (filter_size_ -> size_trans)
            auto size_trans = filter_size_ / 2 + 1;
            auto p_trans = ddrf::cuda::make_unique_device<hipfftComplex>(size_trans, n_col_);

            // calculate the distance between the first elements of two successive lines (needed for cuFFT)
            auto p_exp_dist = static_cast<int>(p_exp.pitch() / sizeof(float));
            auto p_trans_dist = static_cast<int>(p_trans.pitch() / sizeof(hipfftComplex));

            // set the distance between two successive elements
            constexpr auto p_exp_stride = 1;
            constexpr auto p_trans_stride = 1;

            // set storage dimensions of data in memory
            auto p_exp_nembed = p_exp_dist;
            auto p_trans_nembed = p_trans_dist;

            // create plans for forward and inverse FFT
            auto forward = ddrf::cufft::plan<HIPFFT_R2C>{rank, &n,
                                                        &p_exp_nembed, p_exp_stride, p_exp_dist,
                                                        &p_trans_nembed, p_trans_stride, p_trans_dist,
                                                        batch};

            auto inverse = ddrf::cufft::plan<HIPFFT_C2R>{rank, &n,
                                                        &p_trans_nembed, p_trans_stride, p_trans_dist,
                                                        &p_exp_nembed, p_exp_stride, p_exp_dist,
                                                        batch};

            ddrf::cuda::synchronize_stream();

            BOOST_LOG_TRIVIAL(debug) << "Filter setup on device #" << device_ << " completed.";

            while(true)
            {
                auto p = input_();
                if(!p.valid)
                    break;

                ddrf::cuda::launch(p.width, p.height, check, static_cast<const float*>(p.ptr.get()), p.width, p.height, p.ptr.pitch());

                // expand and transform the projection
                expand(p, p_exp, filter_size_, n_col_);
                transform(p_exp.get(), p_trans.get(), forward, p.stream);

                // apply the filter to the transformed projection
                apply_filter(p_trans.get(), k.get(), filter_size_, n_col_, p_trans.pitch(), p.stream);

                // inverse transformation
                transform(p_trans.get(), p_exp.get(), inverse, p.stream);

                // shrink to original size and normalize
                shrink(p_exp, p);
                normalize(p, filter_size_);

                // done
                ddrf::cuda::synchronize_stream(p.stream);
                output_(std::move(p));
            }

            output_(output_type{});
            BOOST_LOG_TRIVIAL(info) << "All projections have been filtered.";
        }
        catch(const ddrf::cuda::bad_alloc& ba)
        {
            BOOST_LOG_TRIVIAL(fatal) << "filter_stage::run() encountered a bad_alloc: " << ba.what();
            throw sre;
        }
        catch(const ddrf::cuda::invalid_argument& ia)
        {
            BOOST_LOG_TRIVIAL(fatal) << "filter_stage::run() passed an invalid argument to the CUDA runtime: " << ia.what();
            throw sre;
        }
        catch(const ddrf::cuda::runtime_error& re)
        {
            BOOST_LOG_TRIVIAL(fatal) << "filter_stage::run() caused a CUDA runtime error: " << re.what();
            throw sre;
        }
        catch(const ddrf::cufft::bad_alloc& ba)
        {
            BOOST_LOG_TRIVIAL(fatal) << "filter_stage::run() encountered a bad allocation in cuFFT: " << ba.what();
            throw sre;
        }
        catch(const ddrf::cufft::invalid_argument& ia)
        {
            BOOST_LOG_TRIVIAL(fatal) << "filter_stage::run() passed an invalid argument to cuFFT: " << ia.what();
            throw sre;
        }
        catch(const ddrf::cufft::runtime_error& re)
        {
            BOOST_LOG_TRIVIAL(fatal) << "filter_stage::run() encountered a cuFFT runtime error: " << re.what();
            throw sre;
        }
    }

    auto filter_stage::set_input_function(std::function<input_type(void)> input) noexcept -> void
    {
        input_ = input;
    }

    auto filter_stage::set_output_function(std::function<void(output_type)> output) noexcept -> void
    {
        output_ = output;
    }
}
