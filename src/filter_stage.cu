#include "hip/hip_runtime.h"
/*
 * This file is part of the ddafa reconstruction program.
 *
 * Copyright (C) 2016 Helmholtz-Zentrum Dresden-Rossendorf
 *
 * Licensed under the EUPL, Version 1.1 or - as soon they will be approved by
 * the European Commission - subsequent version of the EUPL (the "Licence");
 * You may not use this work except in compliance with the Licence.
 * You may obtain a copy of the Licence at:
 *
 * http://ec.europa.eu/idabc/eupl
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the Licence is distributed on an "AS IS" basis,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the Licence for the specific language governing permissions and
 * limitations under the Licence.
 *
 * Date: 18 August 2016
 * Authors: Jan Stephan
 */

#include <algorithm>
#include <atomic>
#include <cmath>
#include <cstddef>
#include <cstdint>
#include <future>
#include <map>
#include <thread>
#include <utility>
#include <vector>

#include <boost/log/trivial.hpp>

#include <hipfft/hipfft.h>

#include <ddrf/cuda/algorithm.h>
#include <ddrf/cuda/coordinates.h>
#include <ddrf/cuda/launch.h>
#include <ddrf/cuda/memory.h>
#include <ddrf/cuda/sync_policy.h>
#include <ddrf/cufft/plan.h>

#include "exception.h"
#include "filter_stage.h"

namespace ddafa
{
    namespace kernel
    {
        __global__ void create_filter(float* __restrict__ r, const std::int32_t* __restrict__ j, std::size_t size, float tau)
        {
            auto x = ddrf::cuda::coord_x();

            /*
             * r(j) with j = [ -(filter_length - 2) / 2, ..., 0, ..., filter_length / 2 ]
             * tau = horizontal pixel distance
             *
             *          1/8 * 1/(tau^2)                     for j = 0
             * r(j) = { 0                                   for even j
             *          -(1 / (2 * j^2 * pi^2 * tau^2))     for odd j
             */
            if(x < size)
            {
                if(j[x] == 0) // does j = 0?
                    r[x] = (1.f / 8.f) * (1.f / powf(tau, 2.f));
                else // j != 0
                {
                    if(j[x] % 2 == 0) // is j even?
                        r[x] = 0.f;
                    else // j is odd
                        r[x] = -(1.f / (2.f * powf(j[x], 2.f) * powf(M_PI, 2.f) * powf(tau, 2.f)));
                }
            }
        }

        __global__ void create_k(hipfftComplex* __restrict__ data, std::size_t filter_size, float tau)
        {
            auto x = ddrf::cuda::coord_x();
            if(x < filter_size)
            {
                auto result = tau * fabsf(sqrtf(powf(data[x].x, 2.f) + powf(data[x].y, 2.f)));
                data[x].x = result;
                data[x].y = result;
            }
        }

        __global__ void apply_filter(hipfftComplex* __restrict__ data, const hipfftComplex* __restrict__ filter,
                                        std::size_t filter_size, std::size_t data_height, std::size_t pitch)
        {
            auto x = ddrf::cuda::coord_x();
            auto y = ddrf::cuda::coord_y();

            if((x < filter_size) && (y < data_height))
            {
                auto row = reinterpret_cast<hipfftComplex*>(reinterpret_cast<char*>(data) + y * pitch);

                row[x].x *= filter[x].x;
                row[x].y *= filter[x].y;
            }
        }
    }

    filter_stage::filter_stage(std::uint32_t n_row, std::uint32_t n_col, float l_px_row)
    : filter_length_{2 * std::pow(2, std::ceil(std::log2(n_row)))}
    , n_col_{n_col}
    , tau_{l_px_row}
    {
        auto err = hipGetDeviceCount(&devices_);
        if(err != hipSuccess)
        {
            BOOST_LOG_TRIVIAL(fatal) << "filter_stage::filter_stage() could not obtain CUDA devices: " << hipGetErrorString(err);
            throw stage_construction_error{"filter_stage::filter_stage() failed"};
        }

        auto filter_futures = std::vector<std::future>{};
        for(auto i = 0; i < devices_; ++i)
            filter_futures.emplace_back(std::async(std::launch::async, &filter_stage::create_filter, this, i));

        try
        {
            for(auto&& f : filter_futures)
                f.get();
        }
        catch(const stage_construction_error& sce)
        {
            BOOST_LOG_TRIVIAL(fatal) << "filter_stage::filter_stage() could not create filters: " << sce.what();
            throw stage_construction_error{"filter_stage::filter_stage() failed"};
        }
    }

    auto filter_stage::run() -> void
    {
        std::map<int, std::future<void>> futures;
        for(int i = 0; i < devices_; ++i)
            futures[i] = std::async(std::launch::async, &filter_stage::process, this, i);

        while(true)
        {
            auto proj = input_();
            safe_push(std::move(proj));
        }

        try
        {
            for(auto&& fp : futures)
                fp.second.get();
        }
        catch(const stage_runtime_error& sre)
        {
            BOOST_LOG_TRIVIAL(fatal) << "filter_stage::run() failed to execute: " << sre.what();
            throw stage_runtime_error{"filter_stage::run() failed"};
        }

        output_(output_type());
        BOOST_LOG_TRIVIAL(info) << "All projections have been filtered.";
    }

    auto filter_stage::set_input_function(std::function<input_type(void)> input) noexcept -> void
    {
        input_ = input;
    }

    auto filter_stage::set_output_function(std::function<void(output_type)> output) noexcept -> void
    {
        output_ = output;
    }

    auto filter_stage::safe_push(input_type proj) -> void
    {
        while(lock_.test_and_set(std::memory_order_acquire))
            std::this_thread::yield();

        if(proj.second.valid)
            input_map_[proj.second.device].push(std::move(proj));
        else
        {
            for(auto i = 0; i < devices; ++i)
                input_map_[i].push(input_type());
        }

        lock_.clear(std::memory_order_release);
    }

    auto filter_stage::safe_pop(int device) -> input_type
    {
        while(input_map_.count(device) == 0)
            std::this_thread::yield();

        while(lock_.test_and_set(std::memory_order_acquire))
            std::this_thread::yield();

        auto& queue = input_map_[device];
        if(queue.empty())
        {
            lock_.clear(std::memory_order_release);
            continue;
        }
        auto proj = std::move(queue.front());
        queue.pop();

        lock_.clear(std::memory_order_release);

        return proj;
    }

    auto filter_stage::create_filter(int device) -> void
    {
        auto err = hipSetDevice(device);
        if(err != hipSuccess)
        {
            BOOST_LOG_TRIVIAL(fatal) << "filter_stage::create_filter() could not set CUDA device: " << hipGetErrorString(err);
            throw stage_construction_error{"filter_stage::create_filter() failed"};
        }

        try
        {
            /*
             * for a more detailed description see kernel::create_filter()
             */
            // create j on the host and fill it with values from -(filter_size_ - 2) / 2 to filter_size / 2
            auto h_j = ddrf::cuda::make_unique_host<std::int32_t>(filter_size_);
            auto size = static_cast<std::int32_t>(filter_size_);
            auto j = (size - 2) / 2;
            std::iota(h_j.get(), h_j.get() + filter_size_, j);

            // create j on the device and copy j from the host to the device
            auto d_j = ddrf::cuda::make_unique_device<float>(filter_size_);
            ddrf::cuda::copy(ddrf::cuda::async, d_j, h_j, filter_size_);

            // create r on the device and calculate the filter values
            auto d_r = ddrf::cuda::make_unique_device<float>(filter_size_);
            ddrf::cuda::launch(filter_size_, kernel::create_filter, d_r.get(), static_cast<const std::int32_t*>(d_j.get()), filter_size_, tau_);

            // move to filter container
            rs_[device] = std::move(d_r);
        }
        catch(const ddrf::cuda::bad_alloc& ba)
        {
            BOOST_LOG_TRIVIAL(fatal) << "filter_stage::create_filter() encountered bad_alloc: " << ba.what();
            throw stage_construction_error{"filter_stage::create_filter() failed"};
        }
        catch(const ddrf::cuda::runtime_error& re)
        {
            BOOST_LOG_TRIVIAL(fatal) << "filter_stage::create_filter() encountered runtime_error: " << re.what();
            throw stage_construction_error{"filter_stage::create_filter() failed"};
        }
        catch(const ddrf::cuda::invalid_argument& ia)
        {
            BOOST_LOG_TRIVIAL(fatal) << "filter_stage::create_filter() encountered invalid_argument: " << ia.what();
            throw stage_construction_error{"filter_stage::create_filter() failed"};
        }
    }

    auto filter_stage::process(int device) -> void
    {
        auto err = hipSetDevice(device);
        if(err != hipSuccess)
        {
            BOOST_LOG_TRIVIAL(fatal) << "filter_stage::process() could not set CUDA device: " << hipGetErrorString(err);
            throw stage_runtime_error{"filter_stage::process() failed"};
        }

        // allocate memory for projection conversion and transformation
        auto converted_proj = ddrf::cuda::pitched_device_ptr<float>{nullptr};
        auto transformed_proj = ddrf::cuda::pitched_device_ptr<hipfftComplex>{nullptr};

        auto transformed_filter = ddrf::cuda::device_ptr<hipfftComplex>{nullptr};
        auto transformed_filter_size = filter_size_ / 2 + 1;
        try
        {
            converted_proj = ddrf::cuda::make_unique_device<float>(filter_size_, n_col_);
            transformed_proj = ddrf::cuda::make_unique_device<hipfftComplex(transformed_filter_size, n_col_);
            transformed_filter_ = ddrf::cuda::make_unique_device<hipfftComplex>(transformed_filter_size);

            ddrf::cuda::fill(ddrf::cuda::async, converted_proj, 0, filter_size_, n_col_);
        }
        catch(const ddrf::cuda::bad_alloc& ba)
        {
            BOOST_LOG_TRIVIAL(fatal) << "filter_stage::process() encountered bad_alloc: " << ba.what();
            throw stage_runtime_error{"filter_stage::process() failed"};
        }
        catch(const ddrf::cuda::invalid_argument& ia)
        {
            BOOST_LOG_TRIVIAL(fatal) << "filter_stage::process() encountered invalid_argument: " << ia.what();
            throw stage_runtime_error{"filter_stage::process() failed"};
        }

        // set up cuFFT
        auto converted_proj_plan = ddrf::cufft::plan<HIPFFT_R2C>{};
        auto filter_plan = ddrf::cufft::plan<HIPFFT_R2C>{};
        auto inverse_plan = ddrf::cufft::plan<HIPFFT_C2R>{};

        auto proj_n = int{filter_size_};
        auto proj_dist = int{converted_proj.pitch() / sizeof(float)};
        auto proj_nembed = proj_dist;

        auto trans_dist = int{transformed_proj.pitch() / sizeof(hipfftComplex)};
        auto trans_nembed = trans_dist;

        try
        {
            converted_proj_plan = ddrf::cufft::plan<HIPFFT_R2C>{1, &proj_n, &proj_nembed, 1, proj_dist, trans_nembed, 1, trans_dist, n_col_};
            filter_plan = ddrf::cufft::plan<HIPFFT_R2C>{proj_n};
            inverse_plan = ddrf::cufft::plan<HIPFFT_C2R>{1, &proj_n, &trans_nembed, 1, trans_dist, &proj_nembed, 1, proj_dist, n_col_};
        }
        catch(const ddrf::cufft::bad_alloc& ba)
        {
            BOOST_LOG_TRIVIAL(fatal) << "filter_stage::process() encountered a bad allocation in cuFFT: " << ba.what();
            throw stage_runtime_error{"filter_stage::process() failed"};
        }

        while(true)
        {
            auto proj = safe_pop(device);
            if(!proj.second.valid)
                break;

            try
            {
                // copy projection to larger projection which has a width of 2^x
                ddrf::cuda::copy(ddrf::cuda::async, converted_proj, proj.first, proj.second.width, proj.second.height);

                // execute the FFT for the projection and the filter
                converted_proj_plan.execute(converted_proj.get(), transformed_proj.get());
                filter_plan.execute(rs_[device].get(), transformed_filter.get());

                // create K
                ddrf::cuda::launch(transformed_filter_size, create_k, transformed_filter.get(), transformed_filter_size, tau_);

                // apply the transformed filter to the transformed projection
                ddrf::cuda::launch(transformed_filter_size, n_col_,
                                    apply_filter,
                                    transformed_proj.get(), transformed_filter.get(), transformed_filter_size, n_col_, transformed_proj.pitch());

                // run inverse FFT on the transformed projection
                inverse_plan.execute(transformed_proj.get(), converted_proj.get());

                // copy back to original projection dimensions
                ddrf::cuda::copy(ddrf::cuda::async, proj.first, converted_proj, proj.second.width, proj.second.height);

                output_(std::move(proj));
            }
            catch(const ddrf::cufft::bad_alloc& ba)
            {
                BOOST_LOG_TRIVIAL(fatal) << "filter_stage::process() encountered a bad allocation in cuFFT: " << ba.what();
                throw stage_runtime_error{"filter_stage::process() failed"};
            }
            catch(const ddrf::cufft::invalid_argument& ia)
            {
                BOOST_LOG_TRIVIAL(fatal) << "filter_stage::process() passed an invalid argument to cuFFT: " << ia.what();
                throw stage_runtime_error{"filter_stage::process() failed"};
            }
            catch(const ddrf::cufft::runtime_error& re)
            {
                BOOST_LOG_TRIVIAL(fatal) << "filter_stage::process() encountered a cuFFT runtime error: " << re.what();
                throw stage_runtime_error{"filter_stage::process() failed"};
            }
            catch(const ddrf::cuda::invalid_argument& ia)
            {
                BOOST_LOG_TRIVIAL(fatal) << "filter_stage::process() passed an invalid argument to the CUDA runtime: " << ia.what();
                throw stage_runtime_error{"filter_stage::process() failed"};
            }
        }
    }
}
