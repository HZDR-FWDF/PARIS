/*
 * CUDADeleter.cu
 *
 *  Created on: 19.11.2015
 *      Author: Jan Stephan
 *
 *      A custom deleter for CUDA memory that is managed by smart pointers. Implementation file.
 */

#include <stdexcept>
#include <string>

#include "CUDADeleter.h"

namespace ddafa
{
	namespace impl
	{
		void CUDADeleter::operator()(void *p)
		{
			hipError_t err = hipFree(p);
			if(err != hipSuccess)
				throw std::runtime_error("CUDADeleter::operator(): " + std::string(hipGetErrorString(err)));
		}
	}
}

