#include "hip/hip_runtime.h"
/*
 * CUDAFilter.cu
 *
 *  Created on: 03.12.2015
 *      Author: Jan Stephan
 *
 *      CUDAFilter takes a weighted projection and applies a filter to it. Implementation file.
 */

#include <cmath>
#include <cstddef>
#include <ctgmath>
#ifdef DDAFA_DEBUG
#include <iostream>
#endif
#include <stdexcept>
#include <string>
#include <thread>
#include <utility>
#include <vector>

#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>

#include "../common/Geometry.h"

#include "CUDAAssert.h"
#include "CUDACommon.h"
#include "CUDADeviceDeleter.h"
#include "CUDAFilter.h"

namespace ddafa
{
	namespace impl
	{
		__global__ void createFilter(float* __restrict__ r, const int* __restrict__ j,
				std::size_t size, float tau)
		{
			int x = getX();

			/*
			 * r(j) with j = [ -(filter_length - 2)/2, ..., 0, ..., filter_length/2 ]
			 * tau = horizontal pixel distance
			 *
			 * 			1/8 * 1/tau^2						j = 0
			 * r(j) = {	0									j even
			 * 			-(1 / (2 * j^2 * pi^2 * tau^2))		j odd
			 *
			 */
			if(x < size)
			{
				if(j[x] == 0) // is j = 0?
					r[x] = (1.f / 8.f) * (1.f / powf(tau, 2)); // j = 0
				else // j != 0
				{
					if(j[x] % 2 == 0) // is j even?
						r[x] = 0.f; // j is even
					else // j is odd
						r[x] = (-1.f / (2.f * powf(j[x], 2) * powf(M_PI, 2) * powf(tau, 2)));

				}
			}
			__syncthreads();
		}

		__global__ void convertProjection(float* __restrict__ output, const float* __restrict__ input,
				unsigned int width, unsigned int height, std::size_t filter_length)
		{
			int x = getX();
			int y = getY();

			if((x < filter_length) && (y < height))
			{
				int idx = x + y * width;
				if(x < width)
					output[idx] = input[idx];
				else
					output[idx] = 0.0f;
			}
			__syncthreads();
		}

		__global__ void convertFiltered(float* __restrict__ output, const float* __restrict__ input,
				unsigned int width, unsigned int height, std::size_t filter_length)
		{
			int x = getX();
			int y = getY();

			if((x < width) && (y < height)) {
				int idx = x + y * width;
				output[idx] = input[idx] / filter_length;
			}
			__syncthreads();
		}

		__global__ void createK(hipfftComplex* __restrict__ data, std::size_t filter_length, float tau)
		{
			int x = getX();
			if(x < filter_length)
			{
				float result = tau * fabsf(sqrtf(powf(data[x].x, 2.f) + powf(data[x].y, 2.f)));
				data[x].x = result;
				data[x].y = result;
			}
			__syncthreads();
		}

		__global__ void applyFilter(hipfftComplex* __restrict__ data, const hipfftComplex* __restrict__ filter,
				std::size_t filter_length, std::uint32_t data_height)
		{
			int x = getX();
			int y = getY();

			if((x < filter_length) && (y < data_height))
			{
				int idx = x + y * filter_length;

				float a1, b1, k1, k2;
				a1 = data[idx].x;
				b1 = data[idx].y;
				k1 = filter[x].x;
				k2 = filter[x].y;

				data[idx].x = a1 * k1;
				data[idx].y = b1 * k2;
			}
			__syncthreads();
		}

		CUDAFilter::CUDAFilter(const ddafa::common::Geometry& geo)
		: filter_length_{static_cast<decltype(filter_length_)>(
				2 * std::pow(2, std::ceil(std::log2(float(geo.det_pixels_column))))
				)}
		, tau_{geo.det_pixel_size_horiz}
		{
			assertCuda(hipGetDeviceCount(&devices_));

			rs_.resize(devices_);

			std::vector<std::thread> filter_creation_threads;
			for(int i = 0; i < devices_; ++i)
			{
				filter_creation_threads.emplace_back(&CUDAFilter::filterProcessor, this, i);
			}

			for(auto&& t : filter_creation_threads)
				t.join();
		}

		CUDAFilter::~CUDAFilter()
		{
		}

		void CUDAFilter::process(CUDAFilter::input_type&& img)
		{
			if(!img.valid())
			{
				// received poisonous pill, time to die
				finish();
				return;
			}

			for(int i = 0; i < devices_; ++i)
			{
				if(img.device() == i)
					processor_threads_.emplace_back(&CUDAFilter::processor, this, std::move(img), i);
			}
		}

		CUDAFilter::output_type CUDAFilter::wait()
		{
			return results_.take();
		}

		void CUDAFilter::filterProcessor(int device)
		{
			assertCuda(hipSetDevice(device));
#ifdef DDAFA_DEBUG
			std::cout << "CUDAFilter: Creating filter on device #" << device << std::endl;
#endif
			float* buffer_raw;
			assertCuda(hipMalloc(&buffer_raw, filter_length_ * sizeof(float)));
			std::unique_ptr<float[], CUDADeviceDeleter> buffer(buffer_raw);

			// see documentation in kernel createFilter for explanation
			std::int32_t j_host_buffer[filter_length_];
			auto filter_length_signed = static_cast<std::int32_t>(filter_length_);
			std::int32_t j = -((filter_length_signed - 2) / 2);
			for(std::size_t k = 0; k <= (filter_length_); ++k, ++j)
				j_host_buffer[k] = j;

			std::int32_t* j_dev_buffer_raw;
			assertCuda(hipMalloc(&j_dev_buffer_raw, filter_length_ * sizeof(std::int32_t)));
			std::unique_ptr<std::int32_t[], CUDADeviceDeleter> j_dev_buffer(j_dev_buffer_raw);
			assertCuda(hipMemcpy(j_dev_buffer.get(), j_host_buffer, filter_length_ * sizeof(std::int32_t),
									hipMemcpyHostToDevice));

			launch1D(filter_length_,
					createFilter,
					buffer.get(), static_cast<const std::int32_t*>(j_dev_buffer.get()),
					filter_length_, tau_);
			rs_[device] = std::move(buffer);
		}

		void CUDAFilter::processor(CUDAFilter::input_type&& img, int device)
		{
			assertCuda(hipSetDevice(device));

#ifdef DDAFA_DEBUG
			std::cout << "CUDAFilter: processing on device #" << device << std::endl;
#endif
			// convert projection to new dimensions
			float* converted_raw;
			assertCuda(hipMalloc(&converted_raw, sizeof(float) * filter_length_ * img.height()));
			std::unique_ptr<float, CUDADeviceDeleter> converted(converted_raw);
			launch2D(filter_length_, img.height(), convertProjection, converted.get(),
					static_cast<const float*>(img.data()), img.width(), img.height(), filter_length_);

			// allocate memory
			std::size_t transformed_filter_length = filter_length_ / 2 + 1; // filter_length_ is always a power of 2
			hipfftComplex* transformed_raw;
			assertCuda(hipMalloc(&transformed_raw,
					sizeof(hipfftComplex) * transformed_filter_length * img.height()));
			std::unique_ptr<hipfftComplex, CUDADeviceDeleter> transformed(transformed_raw);

			hipfftComplex* filter_raw;
			assertCuda(hipMalloc(&filter_raw,
					sizeof(hipfftComplex) * transformed_filter_length));
			std::unique_ptr<hipfftComplex, CUDADeviceDeleter> filter(filter_raw);

			// set up cuFFT
			int n_proj[] = { static_cast<int>(filter_length_) };
			int n_inverse[] = { static_cast<int>(transformed_filter_length) };

			hipfftHandle projectionPlan;
			assertCufft(hipfftCreate(&projectionPlan));
			assertCufft(hipfftSetStream(projectionPlan, 0));
			std::size_t projWorkSize;
			assertCufft(hipfftMakePlanMany(projectionPlan, 1, n_proj, n_proj, 1, filter_length_,
										n_inverse, 1, transformed_filter_length, HIPFFT_R2C,
										img.height(), &projWorkSize));

			hipfftHandle filterPlan;
			assertCufft(hipfftCreate(&filterPlan));
			assertCufft(hipfftSetStream(filterPlan, 0));
			std::size_t filterWorkSize;
			assertCufft(hipfftMakePlan1d(filterPlan, filter_length_, HIPFFT_R2C, 1, &filterWorkSize));

			hipfftHandle inversePlan;
			assertCufft(hipfftCreate(&inversePlan));
			assertCufft(hipfftSetStream(inversePlan, 0));
			std::size_t inverseWorkSize;
			assertCufft(hipfftMakePlanMany(inversePlan, 1, n_proj, n_inverse, 1, transformed_filter_length,
										n_proj, 1, filter_length_, HIPFFT_C2R, img.height(), &inverseWorkSize));

			// run the FFT for projection and filter -- note that R2C transformations are implicitly forward
			assertCufft(hipfftExecR2C(projectionPlan, static_cast<hipfftReal*>(converted.get()), transformed.get()));
			assertCufft(hipfftExecR2C(filterPlan, static_cast<hipfftReal*>(rs_[device].get()), filter.get()));

			// create K
			launch1D(transformed_filter_length, createK, filter.get(), transformed_filter_length, tau_);

			// multiply the results
			launch2D(transformed_filter_length, img.height(), applyFilter, transformed.get(),
				static_cast<const hipfftComplex*>(filter.get()), transformed_filter_length, img.height());

			// run inverse FFT -- note that C2R transformations are implicitly inverse
			assertCufft(hipfftExecC2R(inversePlan, transformed.get(), static_cast<hipfftReal*>(converted.get())));

			// convert back to image dimensions and normalize
			launch2D(filter_length_, img.height(), convertFiltered, img.data(),
					static_cast<const float*>(converted.get()),	img.width(), img.height(), filter_length_);

			// clean up
			assertCufft(hipfftDestroy(inversePlan));
			assertCufft(hipfftDestroy(filterPlan));
			assertCufft(hipfftDestroy(projectionPlan));

			assertCuda(hipStreamSynchronize(0));

			results_.push(std::move(img));
		}

		void CUDAFilter::finish()
		{
#ifdef DDAFA_DEBUG
				std::cout << "CUDAFilter: Received poisonous pill, called finish()" << std::endl;
#endif

				for(auto&& t : processor_threads_)
					t.join();

				results_.push(output_type());
		}
	}
}
