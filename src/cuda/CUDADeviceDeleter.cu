/*
 * CUDADeleter.cu
 *
 *  Created on: 19.11.2015
 *      Author: Jan Stephan
 *
 *      A custom deleter for CUDA device memory that is managed by smart pointers. Implementation file.
 */

#include <stdexcept>
#include <string>

#include "CUDAAssert.h"
#include "CUDADeviceDeleter.h"

namespace ddafa
{
	namespace impl
	{
		auto CUDADeviceDeleter::operator()(void* p) -> void
		{
				assertCuda(hipFree(p));
		}
	}
}
