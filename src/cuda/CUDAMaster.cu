/*
 * CUDAMaster.cu
 *
 *  Created on: 12.11.2015
 *      Author: Jan Stephan
 *
 *      CUDA implementation policy for the Master class. Implementation file.
 */

#include <iostream>
#include <stdexcept>
#include <string>

#include "CUDAMaster.h"

namespace ddafa
{
	namespace impl
	{
		CUDAMaster::CUDAMaster(int device_num)
		: device_{device_num}, number_of_workers_{1}
		{
			hipDeviceProp_t properties;
			hipError_t err = hipGetDeviceProperties(&properties, device_);

			if(err != hipSuccess)
				throw std::runtime_error("CUDAMaster: Invalid device #" + std::to_string(device_));

			if(properties.concurrentKernels == 0)
				std::cout << "CUDAMaster: WARNING: Device #" << device_ << " does not support concurrent kernels."
							<< std::endl;
			else
				std::cout << "CUDAMaster: Device #" << device_ << " supports concurrent kernels." << std::endl;

			// this is ridiculous but CUDA doesn't supply us with the number of resident grids per device
			switch(properties.major)
			{
				case 2:
					number_of_workers_ = 16;
					break;

				case 3:
					switch(properties.minor)
					{
						case 0:
							number_of_workers_ = 16;
							break;

						case 2:
							number_of_workers_ = 4;
							break;

						case 5:
						case 7:
							number_of_workers_ = 32;
							break;

						default:
							throw std::runtime_error("CUDAMaster: Unsupported Compute Capability on device #"
														+ std::to_string(device_)
														+ " (Compute Capability is 3."
														+ std::to_string(properties.minor) + ")");
					}
					break;

				case 5:
					switch(properties.minor)
					{
						case 0:
						case 2:
							number_of_workers_ = 32;
							break;

						case 3:
							number_of_workers_ = 16;
							break;

						default:
							throw std::runtime_error("CUDAMaster: Unsupported Compute Capability on device #"
														+ std::to_string(device_)
														+ " (Compute Capability is 5."
														+ std::to_string(properties.minor) + ")");
					}
					break;

				default:
						throw std::runtime_error("CUDAMaster: Unsupported Compute Capability on device #"
														+ std::to_string(device_)
														+ " (Compute Capability is "
														+ std::to_string(properties.major) + "."
														+ std::to_string(properties.minor) + ")");
			}

			std::cout << "CUDAMaster: Device #" << device_ << " supports " << number_of_workers_
					<< " concurrent kernels." << std::endl;

			std::cout << "CUDAMaster for device #" << device_ << " constructed." << std::endl;
		}

		CUDAMaster::CUDAMaster(CUDAMaster&& other)
		: device_{other.device_}, number_of_workers_{other.number_of_workers_}
		{
		}

		CUDAMaster::~CUDAMaster()
		{
			std::cout << "CUDAMaster for device #" << device_ << " destructed." << std::endl;
		}

		void CUDAMaster::start()
		{
			hipError_t err = hipSetDevice(device_); // bind device to current thread
			if(err != hipSuccess)
			{
				switch(err)
				{
					case hipErrorInvalidDevice:
						throw std::runtime_error("CUDAMaster: Invalid device #" + std::to_string(device_));

					case hipErrorContextAlreadyInUse:
						throw std::runtime_error("CUDAMaster: Device #" + std::to_string(device_)
													+ " already in use");

					default:
						throw std::runtime_error("CUDAMaster: Unknown error while binding device #" +
													std::to_string(device_) + " to current thread.");
				}
			}
		}

		void CUDAMaster::stop()
		{

		}

		int CUDAMaster::workerCount() const noexcept
		{
			return number_of_workers_;
		}

		ddafa::master_worker::Task<CUDAMaster::task_type>
		CUDAMaster::createTask(const ddafa::image::Image* img_ptr)
		{
			std::cout << "CUDAMaster: STUB: createTask() called" << std::endl;
			return ddafa::master_worker::Task<task_type>(0, nullptr, nullptr);
		}
	}
}
