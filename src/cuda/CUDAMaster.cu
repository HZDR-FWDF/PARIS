#include "hip/hip_runtime.h"
/*
 * CUDAMaster.cu
 *
 *  Created on: 12.11.2015
 *      Author: Jan Stephan
 *
 *      CUDA implementation policy for the Master class. Implementation file.
 */

#include <iostream>
#include <stdexcept>
#include <string>

#define BOOST_ALL_DYN_LINK
#include <boost/log/trivial.hpp>

#include "CUDAAssert.h"
#include "CUDAMaster.h"

namespace ddafa
{
	namespace impl
	{
		CUDAMaster::CUDAMaster(int device_num)
		: device_{device_num}, number_of_workers_{1}
		{
			auto properties = hipDeviceProp_t{};
			assertCuda(hipGetDeviceProperties(&properties, device_));

			if(properties.concurrentKernels == 0)
				BOOST_LOG_TRIVIAL(warning) << "CUDAMaster: Device #" << device_ << " does not support concurrent kernels.";
			else
				BOOST_LOG_TRIVIAL(debug) << "CUDAMaster: Device #" << device_ << " supports concurrent kernels.";

			// this is ridiculous but CUDA doesn't supply us with the number of resident grids per device
			switch(properties.major)
			{
				case 2:
					number_of_workers_ = 16;
					break;

				case 3:
					switch(properties.minor)
					{
						case 0:
							number_of_workers_ = 16;
							break;

						case 2:
							number_of_workers_ = 4;
							break;

						case 5:
						case 7:
							number_of_workers_ = 32;
							break;

						default:
							throw std::runtime_error("CUDAMaster: Unsupported Compute Capability on device #"
														+ std::to_string(device_)
														+ " (Compute Capability is 3."
														+ std::to_string(properties.minor) + ")");
					}
					break;

				case 5:
					switch(properties.minor)
					{
						case 0:
						case 2:
							number_of_workers_ = 32;
							break;

						case 3:
							number_of_workers_ = 16;
							break;

						default:
							throw std::runtime_error("CUDAMaster: Unsupported Compute Capability on device #"
														+ std::to_string(device_)
														+ " (Compute Capability is 5."
														+ std::to_string(properties.minor) + ")");
					}
					break;

				default:
						throw std::runtime_error("CUDAMaster: Unsupported Compute Capability on device #"
														+ std::to_string(device_)
														+ " (Compute Capability is "
														+ std::to_string(properties.major) + "."
														+ std::to_string(properties.minor) + ")");
			}

			BOOST_LOG_TRIVIAL(debug) << "CUDAMaster: Device #" << device_ << " supports " << number_of_workers_
					<< " concurrent kernels.";

			BOOST_LOG_TRIVIAL(debug) << "CUDAMaster for device #" << device_ << " constructed.";
		}

		CUDAMaster::CUDAMaster(CUDAMaster&& other)
		: device_{other.device_}, number_of_workers_{other.number_of_workers_}
		{
		}

		CUDAMaster::~CUDAMaster()
		{
			BOOST_LOG_TRIVIAL(debug) << "CUDAMaster for device #" << device_ << " destructed.";
		}

		auto CUDAMaster::start() -> void
		{
			assertCuda(hipSetDevice(device_)); // bind device to current thread
		}

		auto CUDAMaster::stop() -> void
		{
		}

		auto CUDAMaster::workerCount() const noexcept -> int
		{
			return number_of_workers_;
		}

		auto CUDAMaster::createTask(const CUDAMaster::image_type* img_ptr) -> CUDAMaster::task_type
		{
			BOOST_LOG_TRIVIAL(warning) << "CUDAMaster: STUB: createTask() called";
			return task_type(0, nullptr, nullptr);
		}
	}
}
