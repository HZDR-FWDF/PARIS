#include "hip/hip_runtime.h"
/*
 * CUDAWeighting.cu
 *
 *  Created on: 19.11.2015
 *      Author: Jan Stephan
 *
 *      CUDAWeighting manages the concrete implementation of weighting the projections. Implementation file.
 */

#include <cstddef>
#include <cstdint>
#include <stdexcept>
#include <thread>
#include <utility>
#include <vector>

#include "CUDACommon.h"
#include "CUDAWeighting.h"

#include "../image/Image.h"

namespace ddafa
{
	namespace impl
	{
		__global__ void weight(float* img, unsigned width, unsigned height,
								float h_min, float v_min, float d_dist,
								float pixel_size_horiz, float pixel_size_vert)
		{
			int j = blockIdx.x * blockDim.x + threadIdx.x; // row index
			int i = blockIdx.y * blockDim.y + threadIdx.y; // column index

			if((j >= width) || (i >= height))
				return;

			int idx = i + j * width; // current pixel

			// detector coordinates
			float h_j = (pixel_size_horiz / 2) + j * pixel_size_horiz + h_min;
			float v_i = (pixel_size_vert / 2) + i * pixel_size_vert + v_min;

			// calculate weight
			float w_ij = d_dist * rsqrtf(powf(d_dist, 2) + powf(h_j, 2) + powf(v_i, 2));

			// apply
			img[idx] = img[idx] * w_ij;
		}

		CUDAWeighting::CUDAWeighting(ddafa::common::Geometry geo)
		: geo_(geo)
		, h_min_{-geo.det_offset_horiz - ((geo.det_pixels_row * geo.det_pixel_size_horiz) / 2)}
		, v_min_{-geo.det_offset_vert - ((geo.det_pixel_column * geo.det_pixel_size_vert) / 2)}
		, d_dist_{geo.dist_det - geo.dist_src}
		{
			hipError_t err = hipGetDeviceCount(&devices_);

			switch(err)
			{
				case hipSuccess:
					break;

				case hipErrorNoDevice:
					throw std::runtime_error("CUDAWeighting: No CUDA devices found.");

				case hipErrorInsufficientDriver:
					throw std::runtime_error("CUDAWeighting: Insufficient driver.");
			}
		}

		CUDAWeighting::~CUDAWeighting()
		{
		}

		void CUDAWeighting::process(CUDAWeighting::input_type&& img)
		{
			if(!img.valid())
			{
				// received poisonous pill, time to die
				results_.push(output_type());
				return;
			}

			std::vector<std::thread> processor_threads;
			for(int i = 0; i < devices_; ++i)
			{
				// copy image to device
				hipSetDevice(i);
				float* dev_buffer;
				std::size_t size = img.width() * img.height() * sizeof(float);
				hipError_t err = hipMalloc(&dev_buffer, size);

				switch(err)
				{
					case hipErrorOutOfMemory:
						throw std::runtime_error("CUDAWeighting: Error while allocating memory");

					case hipSuccess:
						default:
						break;
				}

				err = hipMemcpy(dev_buffer, img.data(), size, hipMemcpyHostToDevice);
				switch(err)
				{
					case hipErrorInvalidValue:
						throw std::runtime_error("CUDAWeighting: Invalid value");

					case hipErrorInvalidDevicePointer:
						throw std::runtime_error("CUDAWeighting: Invalid device pointer");

					case hipErrorInvalidMemcpyDirection:
						throw std::runtime_error("CUDAWeighting: Invalid memcpy direction");

					case hipSuccess:
						default:
						break;
				}
				// execute kernel
				processor_threads.emplace_back(&CUDAWeighting::processor, this, dev_buffer, size,
												img.width(), img.height());
			}

			for(auto&& t : processor_threads)
				t.join();
		}

		CUDAWeighting::output_type CUDAWeighting::wait()
		{
			return results_.take();
		}

		void CUDAWeighting::processor(float* buffer, std::size_t size, std::uint32_t width, std::uint32_t height)
		{
			launch(size, weight, buffer, width, height, h_min_, v_min_, d_dist_,
					geo_.det_pixel_size_horiz, geo_.det_pixel_size_vert);
			output_type result(width, height, buffer);

			int device;
			hipGetDevice(&device);

			result.setDevice(device);

			results_.push(std::move(result));
		}
	}
}
