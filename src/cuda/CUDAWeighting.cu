#include "hip/hip_runtime.h"
/*
 * CUDAWeighting.cu
 *
 *  Created on: 19.11.2015
 *      Author: Jan Stephan
 *
 *      CUDAWeighting manages the concrete implementation of weighting the projections. Implementation file.
 */

#include <cstddef>
#include <cstdint>
#ifdef DDAFA_DEBUG
#include <iostream>
#endif
#include <stdexcept>
#include <string>
#include <utility>

#include "CUDACommon.h"
#include "CUDADeleter.h"
#include "CUDAWeighting.h"

#include "../common/Geometry.h"
#include "../image/Image.h"

namespace ddafa
{
	namespace impl
	{
		__global__ void weight(float* img, unsigned int width, unsigned int height,
								float h_min, float v_min, float d_dist,
								float pixel_size_horiz, float pixel_size_vert)
		{
			int j = blockIdx.x * blockDim.x + threadIdx.x; // row index
			int i = blockIdx.y * blockDim.y + threadIdx.y; // column index

			if((j < width) && (i < height))
			{
				int idx = j + i * width; // current pixel

				// detector coordinates
				float h_j = (pixel_size_horiz / 2) + j * pixel_size_horiz + h_min;
				float v_i = (pixel_size_vert / 2) + i * pixel_size_vert + v_min;

				// calculate weight
				float w_ij = d_dist * rsqrtf(powf(d_dist, 2) + powf(h_j, 2) + powf(v_i, 2));

				// apply
				img[idx] = img[idx] * w_ij;
			}
			__syncthreads();
		}

		CUDAWeighting::CUDAWeighting(ddafa::common::Geometry&& geo)
		: geo_(geo)
		, h_min_{-geo.det_offset_horiz - ((geo.det_pixels_row * geo.det_pixel_size_horiz) / 2)}
		, v_min_{-geo.det_offset_vert - ((geo.det_pixel_column * geo.det_pixel_size_vert) / 2)}
		, d_dist_{307.5} //FIXME: Remove fixed value
		{
			assertCuda(hipGetDeviceCount(&devices_));
		}

		CUDAWeighting::~CUDAWeighting()
		{
		}

		void CUDAWeighting::process(CUDAWeighting::input_type&& img)
		{
			if(!img.valid())
			{
				// received poisonous pill, time to die
				finish();
				return;
			}

			for(int i = 0; i < devices_; ++i)
			{
#ifdef DDAFA_DEBUG
				std::cout << "CUDAWeighting: Copying to device #" << i << std::endl;
#endif
				// copy image to device
				assertCuda(hipSetDevice(i));
				float* dev_buffer;
				std::size_t size = img.width() * img.height() * sizeof(float);
				assertCuda(hipMalloc(&dev_buffer, size));

#ifdef DDAFA_DEBUG
				std::cout << "CUDAWeighting: Image dimensions are " << img.width()
						<< "x" << img.height() << std::endl;
				std::cout << "Size on device: " << size << " bytes" << std::endl;
#endif

				assertCuda(hipMemcpy(dev_buffer, img.data(), size, hipMemcpyHostToDevice));

				// execute kernel
				processor_threads_.emplace_back(&CUDAWeighting::processor, this, dev_buffer, size,
												img.width(), img.height(), i);
			}
		}

		CUDAWeighting::output_type CUDAWeighting::wait()
		{
			return results_.take();
		}

		void CUDAWeighting::processor(float* buffer, std::size_t size, std::uint32_t width, std::uint32_t height, int device)
		{
			assertCuda(hipSetDevice(device));
#ifdef DDAFA_DEBUG
			std::cout << "CUDAWeighting: processing on device #" << device << std::endl;
#endif

			launch2D(width, height, weight, buffer, width, height, h_min_, v_min_, d_dist_,
					geo_.det_pixel_size_horiz, geo_.det_pixel_size_vert);

			output_type result(width, height, std::unique_ptr<float, CUDADeleter>(buffer));
			result.setDevice(device);

			results_.push(std::move(result));
		}

		void CUDAWeighting::finish()
		{
#ifdef DDAFA_DEBUG
			std::cout << "CUDAWeighting: Received poisonous pill, called finish()" << std::endl;
#endif
			for(auto&& t : processor_threads_)
				t.join();

#ifdef DDAFA_DEBUG
			std::cout << "CUDAWeighting: All threads joined." << std::endl;
#endif

			results_.push(output_type());
		}
	}
}
