/*
 * CUDAFeldkamp.cu
 *
 *  Created on: 12.11.2015
 *      Author: Jan Stephan
 *
 *      This class is the concrete backprojection implementation for the Stage class. Implementation file.
 */

#include <stdexcept>
#include <string>

#include "../image/Image.h"
#include "../master_worker/Master.h"

#include "CUDAAssert.h"
#include "CUDAFeldkamp.h"

namespace ddafa
{
	namespace impl
	{
		CUDAFeldkamp::CUDAFeldkamp()
		{
			int device_count;
			assertCuda(hipGetDeviceCount(&device_count));

			for(int i = 0; i < device_count; ++i)
				masters_.emplace_back(i);

			for(auto&& master : masters_)
				master_threads_.emplace_back(&master_type::start, &master);
		}

		CUDAFeldkamp::~CUDAFeldkamp()
		{
		}

		void CUDAFeldkamp::process(CUDAFeldkamp::input_type&& img)
		{
			// do NOT delete this pointer
			input_type* img_ptr = &img;
			for(auto&& master : masters_)
				master.input(img_ptr);
		}

		CUDAFeldkamp::output_type CUDAFeldkamp::wait()
		{
			for(auto&& thread : master_threads_)
				thread.join();
			return CUDAFeldkamp::output_type();
		}
	}
}
