/*
 * CUDAFeldkamp.cu
 *
 *  Created on: 12.11.2015
 *      Author: Jan Stephan
 *
 *      This class is the concrete backprojection implementation for the Stage class. Implementation file.
 */
#
#include <stdexcept>

#include "../image/Image.h"
#include "../master_worker/Master.h"

#include "CUDAFeldkamp.h"

namespace ddafa
{
	namespace impl
	{
		CUDAFeldkamp::CUDAFeldkamp()
		{
			int device_count;
			hipError_t err = hipGetDeviceCount(&device_count);

			switch(err)
			{
				case hipSuccess:
					break;

				case hipErrorNoDevice:
					throw std::runtime_error("CUDAFeldkamp: No CUDA devices found.");

				case hipErrorInsufficientDriver:
					throw std::runtime_error("CUDAFeldkamp: Insufficient driver.");
			}

			for(int i = 0; i < device_count; ++i)
				masters_.emplace_back(i);

			for(auto&& master : masters_)
				master_threads_.emplace_back(&master_type::start, &master);
		}

		CUDAFeldkamp::~CUDAFeldkamp()
		{
		}

		void CUDAFeldkamp::process(CUDAFeldkamp::input_image_type&& img)
		{
			// do NOT delete this pointer
			input_image_type* img_ptr = &img;
			for(auto&& master : masters_)
				master.input(img_ptr);
		}

		CUDAFeldkamp::output_image_type CUDAFeldkamp::wait()
		{
			for(auto&& thread : master_threads_)
				thread.join();
			return output_image_type();
		}
	}
}
