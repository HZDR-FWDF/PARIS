/*
 * CUDAFeldkamp.cu
 *
 *  Created on: 12.11.2015
 *      Author: Jan Stephan
 *
 *      This class is the concrete backprojection implementation for the Stage class. Implementation file.
 */

#include <stdexcept>
#include <string>

#include "../image/Image.h"
#include "../master_worker/Master.h"

#include "CUDAFeldkamp.h"

namespace ddafa
{
	namespace impl
	{
		CUDAFeldkamp::CUDAFeldkamp()
		{
			int device_count;
			hipError_t err = hipGetDeviceCount(&device_count);
			if(err != hipSuccess)
				throw std::runtime_error("CUDAFeldkamp::CUDAFeldkamp: " + std::string(hipGetErrorString(err)));

			for(int i = 0; i < device_count; ++i)
				masters_.emplace_back(i);

			for(auto&& master : masters_)
				master_threads_.emplace_back(&master_type::start, &master);
		}

		CUDAFeldkamp::~CUDAFeldkamp()
		{
		}

		void CUDAFeldkamp::process(CUDAFeldkamp::input_type&& img)
		{
			// do NOT delete this pointer
			input_type* img_ptr = &img;
			for(auto&& master : masters_)
				master.input(img_ptr);
		}

		CUDAFeldkamp::output_type CUDAFeldkamp::wait()
		{
			for(auto&& thread : master_threads_)
				thread.join();
			return CUDAFeldkamp::output_type();
		}
	}
}
