#include "hip/hip_runtime.h"
/*
 * CUDAWorker.cu
 *
 *  Created on: 12.11.2015
 *      Author: Jan Stephan
 *
 *      CUDA implementation policy for the Worker class. Implementation file.
 */

#define BOOST_ALL_DYN_LINK
#include <boost/log/trivial.hpp>

#include "CUDAWorker.h"

#include "../master_worker/Task.h"

namespace ddafa
{
	namespace impl
	{
		CUDAWorker::~CUDAWorker()
		{
		}

		void CUDAWorker::start()
		{
		}

		CUDAWorker::result_type	CUDAWorker::process(CUDAWorker::task_type&& current_task)
		{
			BOOST_LOG_TRIVIAL(warning) << "CUDAWorker: STUB: process() called";
			return result_type(0, nullptr, nullptr);
		}
	}
}
