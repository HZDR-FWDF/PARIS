#include "hip/hip_runtime.h"
#include <cstddef>
#include <cstdint>
#include <stdexcept>
#include <string>
#include <utility>

#define BOOST_ALL_DYN_LINK
#include <boost/log/trivial.hpp>

#include <ddrf/Image.h>
#include <ddrf/cuda/Check.h>
#include <ddrf/cuda/Coordinates.h>
#include <ddrf/cuda/Launch.h>

#include "Weighting.h"

#include "../common/Geometry.h"

namespace ddafa
{
	namespace cuda
	{
		__global__ void weight(float* img,
								std::size_t width, std::size_t height, std::size_t pitch,
								float h_min, float v_min, float d_dist,
								float pixel_size_horiz, float pixel_size_vert)
		{
			auto j = ddrf::cuda::getX(); // column index
			auto i = ddrf::cuda::getY(); // row index

			if((j < width) && (i < height))
			{
				auto* row = reinterpret_cast<float*>(reinterpret_cast<char*>(img) + i * pitch);

				// detector coordinates
				auto h_j = (pixel_size_horiz / 2) + j * pixel_size_horiz + h_min;
				auto v_i = (pixel_size_vert / 2) + i * pixel_size_vert + v_min;

				// calculate weight
				auto w_ij = d_dist * rsqrtf(powf(d_dist, 2) + powf(h_j, 2) + powf(v_i, 2));

				// apply
				row[j] = row[j] * w_ij;
			}
			__syncthreads();
		}

		Weighting::Weighting(const common::Geometry& geo)
		: geo_(geo)
		, h_min_{-(geo.det_offset_horiz * geo.det_pixel_size_horiz) - ((static_cast<float>(geo.det_pixels_row) * geo.det_pixel_size_horiz) / 2)}
		, v_min_{-(geo.det_offset_vert * geo.det_pixel_size_vert) - ((static_cast<float>(geo.det_pixels_column) * geo.det_pixel_size_vert) / 2)}
		, d_dist_{geo.dist_det + geo.dist_src}
		{
			ddrf::cuda::check(hipGetDeviceCount(&devices_));
		}

		auto Weighting::process(input_type&& img) -> void
		{
			if(!img.valid())
			{
				// received poisonous pill, time to die
				finish();
				return;
			}

			for(auto i = 0; i < devices_; ++i)
			{
				// execute kernel
				processor_threads_.emplace_back(&Weighting::processor, this, img, i);
			}
		}

		auto Weighting::wait() -> output_type
		{
			return results_.take();
		}

		auto Weighting::processor(const input_type& img, int device) -> void
		{
			ddrf::cuda::check(hipSetDevice(device));
			// BOOST_LOG_TRIVIAL(debug) << "CUDAWeighting: processing on device #" << device;

			auto result = output_type{};
			result.setDevice(device);
			result = img;

			ddrf::cuda::launch(result.width(), result.height(),
					weight,
					result.data(), result.width(), result.height(), result.pitch(), h_min_, v_min_, d_dist_,
					geo_.det_pixel_size_horiz, geo_.det_pixel_size_vert);

			ddrf::cuda::check(hipStreamSynchronize(0));
			results_.push(std::move(result));
		}

		auto Weighting::finish() -> void
		{
			BOOST_LOG_TRIVIAL(debug) << "CUDAWeighting: Received poisonous pill, called finish()";

			for(auto&& t : processor_threads_)
				t.join();

			results_.push(output_type());
		}
	}
}
