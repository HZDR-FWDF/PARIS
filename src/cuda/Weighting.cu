#include "hip/hip_runtime.h"
#include <cstddef>
#include <cstdint>
#include <future>
#include <stdexcept>
#include <string>
#include <utility>

#define BOOST_ALL_DYN_LINK
#include <boost/log/trivial.hpp>

#include <ddrf/Image.h>
#include <ddrf/cuda/Check.h>
#include <ddrf/cuda/Coordinates.h>
#include <ddrf/cuda/Launch.h>

#include "Weighting.h"

#include "../common/Geometry.h"

namespace ddafa
{
	namespace cuda
	{
		__global__ void weight(float* img,
								std::size_t width, std::size_t height, std::size_t pitch,
								float h_min, float v_min, float d_dist,
								float pixel_size_horiz, float pixel_size_vert)
		{
			auto j = ddrf::cuda::getX(); // column index
			auto i = ddrf::cuda::getY(); // row index

			if((j < width) && (i < height))
			{
				auto* row = reinterpret_cast<float*>(reinterpret_cast<char*>(img) + i * pitch);

				// detector coordinates
				auto h_j = (pixel_size_horiz / 2) + j * pixel_size_horiz + h_min;
				auto v_i = (pixel_size_vert / 2) + i * pixel_size_vert + v_min;

				// calculate weight
				auto w_ij = d_dist * rsqrtf(powf(d_dist, 2) + powf(h_j, 2) + powf(v_i, 2));

				// apply
				row[j] = row[j] * w_ij;
			}
			__syncthreads();
		}

		Weighting::Weighting(const common::Geometry& geo)
		: geo_(geo)
		, h_min_{-(geo.det_offset_horiz * geo.det_pixel_size_horiz) - ((static_cast<float>(geo.det_pixels_row) * geo.det_pixel_size_horiz) / 2)}
		, v_min_{-(geo.det_offset_vert * geo.det_pixel_size_vert) - ((static_cast<float>(geo.det_pixels_column) * geo.det_pixel_size_vert) / 2)}
		, d_dist_{geo.dist_det + geo.dist_src}
		{
			CHECK(hipGetDeviceCount(&devices_));
			for(auto i = 0; i < devices_; ++i)
			{
				auto pr = std::promise<bool>{};
				processor_futures_[i].emplace_back(pr.get_future());
				pr.set_value(true);
			}
		}

		auto Weighting::process(input_type&& img) -> void
		{
			if(!img.valid())
			{
				// received poisonous pill, time to die
				finish();
				return;
			}

			auto pr = std::promise<bool>{};
			processor_futures_[img.device()].emplace_back(pr.get_future());
			processor_threads_.emplace_back(&Weighting::processor, this, std::move(img), std::move(pr));
		}

		auto Weighting::wait() -> output_type
		{
			return results_.take();
		}

		auto Weighting::processor(input_type&& img, std::promise<bool> pr) -> void
		{
			auto device = img.device();
			auto future = std::move(processor_futures_[device].front());
			processor_futures_[device].pop_front();
			auto start = future.get();
			start = !start;

			CHECK(hipSetDevice(device));
			BOOST_LOG_TRIVIAL(debug) << "cuda::Weighting: processing on device #" << img.device();

			ddrf::cuda::launch(img.width(), img.height(),
					weight,
					img.data(), img.width(), img.height(), img.pitch(), h_min_, v_min_, d_dist_,
					geo_.det_pixel_size_horiz, geo_.det_pixel_size_vert);

			CHECK(hipStreamSynchronize(0));
			results_.push(std::move(img));
			pr.set_value(true);
		}

		auto Weighting::finish() -> void
		{
			BOOST_LOG_TRIVIAL(debug) << "CUDAWeighting: Received poisonous pill, called finish()";

			for(auto&& t : processor_threads_)
				t.join();

			for(auto& kv : processor_futures_)
			{
				for(auto& f : kv.second)
				{
					auto val = f.get();
					val = !val;
				}
			}

			results_.push(output_type());
		}
	}
}
