#include "hip/hip_runtime.h"
/*
 * CUDAFeldkamp.cu
 *
 *  Created on: 12.11.2015
 *      Author: Jan Stephan
 *
 *      This class is the concrete backprojection implementation for the Stage class. Implementation file.
 */

#include <array>
#include <cstddef>
#include <cmath>
#include <thread>
#include <vector>

#include <ddrf/Image.h>
#include <ddrf/cuda/Check.h>
#include <ddrf/cuda/Coordinates.h>
#include <ddrf/cuda/Launch.h>

#include "../common/Geometry.h"
#include "Feldkamp.h"
#include "FeldkampScheduler.h"

namespace ddafa
{
	namespace cuda
	{
		__global__ void init_volume(float* vol, std::size_t width, std::size_t height, std::size_t depth, std::size_t pitch)
		{
			auto x = ddrf::cuda::getX();
			auto y = ddrf::cuda::getY();
			auto z = ddrf::cuda::getZ();

			if((x < width) && (y < height) && (z < depth))
			{
				auto slice_pitch = pitch * height;
				auto slice = reinterpret_cast<char*>(vol) + z * slice_pitch;
				auto row = reinterpret_cast<float*>(slice + y * pitch);

				row[x] = 0.f;
			}
		}

		__global__ void backproject(float *vol, std::size_t vol_w, std::size_t vol_h, std::size_t vol_d, std::size_t vol_pitch,
									const float *proj, std::size_t proj_w, std::size_t proj_h, std::size_t proj_pitch,
									unsigned int i, float angle, float dist_src, float dist_det,
									std::uint32_t num_proj)
		{
			auto x = ddrf::cuda::getX();
			auto y = ddrf::cuda::getY();
			auto z = ddrf::cuda::getZ();

			if((x < vol_w) && (y < vol_h) && (z < vol_d))
			{
				auto slice_pitch = vol_pitch * vol_h;
				auto slice = reinterpret_cast<char*>(vol) + z * slice_pitch;
				auto row = reinterpret_cast<float*>(slice + y * vol_pitch);
				auto proj_row = reinterpret_cast<const float*>(reinterpret_cast<const char*>(proj) + y * proj_pitch); // FIXME: y is wrong here

				auto denominator = dist_src - x * cosf(angle) - y * sinf(angle);
				auto dist_so = dist_src + dist_det;
				auto u = (dist_so * (-x * sinf(angle) + y * cosf(angle))) / denominator;
				auto v = (dist_so * z) / denominator;
				auto w2 = dist_src / denominator;

				// this is wrong
				row[x] += (1.f / (2.f * M_PI * num_proj)) * proj_row[static_cast<std::uint32_t>(u) + static_cast<std::uint32_t>(v) * proj_w] * w2;
			}
		}


		Feldkamp::Feldkamp(const common::Geometry& geo)
		: scheduler_{FeldkampScheduler<float>::instance(geo)}
		, geo_(geo), input_num_{0u}, input_num_set_{false}
		{
			ddrf::cuda::check(hipGetDeviceCount(&devices_));
			std::vector<std::thread> creation_threads;
			for(auto i = 0; i < devices_; ++i)
			{
				creation_threads.emplace_back(&Feldkamp::create_volumes, this, i);
			}

			for(auto&& t : creation_threads)
				t.join();
		}

		auto Feldkamp::process(input_type&& img) -> void
		{
		}

		auto Feldkamp::wait() -> output_type
		{
			return output_type{};
		}

		auto Feldkamp::set_input_num(std::uint32_t num) noexcept -> void
		{
			input_num_ = num;
			input_num_set_ = true;
		}

		auto Feldkamp::create_volumes(int device) -> void
		{
			ddrf::cuda::check(hipSetDevice(device));
			auto volume_num = scheduler_.chunkNumber(device);
			auto dimensions = scheduler_.chunkDimensions(device);
			for(auto i = 0u; i < volume_num; ++i)
			{
				auto first_row = dimensions[i].first;
				auto last_row = dimensions[i].second;
				auto rows = last_row - first_row + 1u;

				auto ptr = ddrf::cuda::make_device_ptr<float>(geo_.det_pixels_row, geo_.det_pixels_row, rows);
				ddrf::cuda::launch(ptr.width(), ptr.height(), ptr.depth(),
									init_volume,
									ptr.get(), ptr.width(), ptr.height(), ptr.depth(), ptr.pitch());

				volume_map_[device].emplace_back(std::move(ptr));
			}
		}
	}
}
