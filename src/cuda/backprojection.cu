#include "hip/hip_runtime.h"
/*
 * This file is part of the PARIS reconstruction program.
 *
 * Copyright (C) 2016 Helmholtz-Zentrum Dresden-Rossendorf
 *
 * PARIS is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * PARIS is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with PARIS. If not, see <http://www.gnu.org/licenses/>.
 *
 * Date: 04 December 2016
 * Authors: Jan Stephan <j.stephan@hzdr.de>
 */

#include <cstddef>
#include <cstdint>

#include <boost/log/trivial.hpp>

#include <glados/cuda/coordinates.h>
#include <glados/cuda/launch.h>
#include <glados/cuda/utility.h>

#include "../exception.h"
#include "../region_of_interest.h"

#include "backend.h"
#include "backprojection_constants.h"

namespace paris
{
    namespace cuda
    {
        namespace
        {
            // note that each device will automatically keep track of its own symbol
            __device__ __constant__ backprojection_constants dev_consts__{};
            __device__ __constant__ region_of_interest dev_roi__{};

            inline __device__ auto vol_centered_coordinate(unsigned int coord,
                                                           std::uint32_t dim, float size)
            -> float
            {
                auto size2 = size / 2.f;
                return -(dim * size2) + size2 + coord * size;
            }

            inline __device__ auto proj_real_coordinate(float coord, std::uint32_t dim,
                                                        float size, float offset) -> float
            {
                auto size2 = size / 2.f;
                auto min = -(dim * size2) - offset;
                return (coord - min) / size - (1.f / 2.f);
            }

            template <bool enable_roi>
            __global__ void backprojection_kernel(float* __restrict__ vol, std::size_t vol_pitch,
                                                  hipTextureObject_t proj, float angle_sin,
                                                  float angle_cos)
            {
                auto k = glados::cuda::coord_x();
                auto l = glados::cuda::coord_y();
                auto m = glados::cuda::coord_z();

                if((k < dev_consts__.vol_dim_x) &&
                   (l < dev_consts__.vol_dim_y) &&
                   (m < dev_consts__.vol_dim_z))
                {
                    auto slice_pitch = vol_pitch * dev_consts__.vol_dim_y;
                    auto slice = reinterpret_cast<char*>(vol) + m * slice_pitch;
                    auto row = reinterpret_cast<float*>(slice + l * vol_pitch);

                    // load old value from global memory while executing other instructions
                    auto old_val = row[k];
                    
                    // add ROI offset. If enable_roi == false, this will be optimized away
                    if(enable_roi)
                    {
                        k += dev_roi__.x1;
                        l += dev_roi__.y1;
                        m += dev_roi__.z1;
                    }

                    // add offset for the current subvolume
                    m += dev_consts__.vol_offset;

                    // get centered coordinates -- volume center at (0, 0, 0)
                    auto x_k = vol_centered_coordinate(k, dev_consts__.vol_dim_x_full,
                                                            dev_consts__.l_vx_x);
                    auto y_l = vol_centered_coordinate(l, dev_consts__.vol_dim_y_full,
                                                            dev_consts__.l_vx_y);
                    auto z_m = vol_centered_coordinate(m, dev_consts__.vol_dim_z_full,
                                                            dev_consts__.l_vx_z);

                    // rotate coordinates
                    auto s = x_k * angle_cos + y_l * angle_sin;
                    auto t = -x_k * angle_sin + y_l * angle_cos;

                    // project rotated coordinates
                    auto factor = dev_consts__.d_sd / (s + dev_consts__.d_so);
                    // add 0.5 to each coordinate to deal with CUDA's filtering mechanism
                    auto h = proj_real_coordinate(t * factor, dev_consts__.proj_dim_x,
                                                                dev_consts__.l_px_x,
                                                                dev_consts__.delta_s) + 0.5f;
                    auto v = proj_real_coordinate(z_m * factor, dev_consts__.proj_dim_y,
                                                                dev_consts__.l_px_y,
                                                                dev_consts__.delta_t) + 0.5f;

                    // get projection value (note the implicit linear interpolation)
                    auto det = tex2D<float>(proj, h, v);

                    // backproject
                    auto u = -(dev_consts__.d_so / (s + dev_consts__.d_so));

                    // restore old coordinate for writing.
                    if(enable_roi)
                        k -= dev_roi__.x1;

                    // write value
                    row[k] = old_val + 0.5f * det * u * u;
                }
            }
        }

        auto backproject(const projection_device_type& p, volume_device_type& v, std::uint32_t v_offset,
                         const detector_geometry& det_geo, const volume_geometry& vol_geo,
                         bool enable_roi, const region_of_interest& roi,
                         float sin, float cos, float delta_s, float delta_t)  -> void
        {
            // constants for the backprojection - these never change
            static const auto v_dim_x_full = vol_geo.dim_x;
            static const auto v_dim_y_full = vol_geo.dim_y;
            static const auto v_dim_z_full = vol_geo.dim_z;

            static const auto l_vx_x = vol_geo.l_vx_x;
            static const auto l_vx_y = vol_geo.l_vx_y;
            static const auto l_vx_z = vol_geo.l_vx_z;

            static const auto p_dim_x = det_geo.n_row;
            static const auto p_dim_y = det_geo.n_col;

            static const auto l_px_x = det_geo.l_px_row;
            static const auto l_px_y = det_geo.l_px_col;

            static const auto d_s = delta_s;
            static const auto d_t = delta_t;

            static const auto d_so = det_geo.d_so;
            static const auto d_sd = std::abs(det_geo.d_so) + std::abs(det_geo.d_od);

            // variable for the backprojection - might change between subvolumes
            thread_local static auto offset = v_offset;

            // local stream
            thread_local static auto s = hip_stream{};

            // initialise device constants
            thread_local static auto consts = backprojection_constants {
                v.dim_x,
                v_dim_x_full,
                v.dim_y,
                v_dim_y_full,
                v.dim_z,
                v_dim_z_full,
                offset,
                l_vx_x,
                l_vx_y,
                l_vx_z,
                p_dim_x,
                p_dim_y,
                l_px_x,
                l_px_y,
                d_s,
                d_t,
                d_so,
                d_sd
            };

            auto err = hipMemcpyToSymbolAsync(HIP_SYMBOL(dev_consts__), &consts, sizeof(consts), 0u, hipMemcpyHostToDevice,
                                               s.stream);
            if(err != hipSuccess)
            {
                BOOST_LOG_TRIVIAL(fatal) << "Could not initialise device constants: " << hipGetErrorString(err);
                throw stage_runtime_error{"backproject() failed"};
            }

            // create a CUDA texture from the projection
            auto res_desc = hipResourceDesc{};
            res_desc.resType = hipResourceTypePitch2D;
            res_desc.res.pitch2D.desc = hipCreateChannelDesc<float>();
            res_desc.res.pitch2D.devPtr = reinterpret_cast<void*>(p.buf.get());
            res_desc.res.pitch2D.width = p.dim_x;
            res_desc.res.pitch2D.height = p.dim_y;
            res_desc.res.pitch2D.pitchInBytes = p.buf.pitch();

            auto tex_desc = hipTextureDesc{};
            tex_desc.addressMode[0] = hipAddressModeBorder;
            tex_desc.addressMode[1] = hipAddressModeBorder;
            tex_desc.filterMode = hipFilterModeLinear;
            tex_desc.readMode = hipReadModeElementType;
            tex_desc.normalizedCoords = 0;

            auto tex = hipTextureObject_t{0};
            err = hipCreateTextureObject(&tex, &res_desc, &tex_desc, nullptr);
            if(err != hipSuccess)
            {
                BOOST_LOG_TRIVIAL(fatal) << "Could not create CUDA texture: " << hipGetErrorString(err);
                throw stage_runtime_error{"backproject() failed"};
            }

            // apply ROI as needed and backproject
            if(enable_roi)
            {
                err = hipMemcpyToSymbolAsync(HIP_SYMBOL(dev_roi__), &roi, sizeof(roi), 0u, hipMemcpyHostToDevice, s.stream);
                if(err != hipSuccess)
                {
                    BOOST_LOG_TRIVIAL(fatal) << "Could not initialise device ROI: " << hipGetErrorString(err);
                    throw stage_runtime_error{"backproject() failed"};
                }

                glados::cuda::launch_async(s.stream, v.dim_x, v.dim_y, v.dim_z, backprojection_kernel<true>,
                                           v.buf.get(), v.buf.pitch(), tex, sin, cos);
            }
            else
                glados::cuda::launch_async(s.stream, v.dim_x, v.dim_y, v.dim_z, backprojection_kernel<false>,
                                           v.buf.get(), v.buf.pitch(), tex, sin, cos);

            glados::cuda::synchronize_stream(s.stream);
            err = hipDestroyTextureObject(tex);
            if(err != hipSuccess)
            {
                BOOST_LOG_TRIVIAL(fatal) << "Could not destroy CUDA texture: " << hipGetErrorString(err);
                throw stage_runtime_error{"backproject() failed"};
            }
        }
    }
}
