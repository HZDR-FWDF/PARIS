#include "hip/hip_runtime.h"
/*
 * This file is part of the ddafa reconstruction program.
 *
 * Copyright (C) 2016 Helmholtz-Zentrum Dresden-Rossendorf
 *
 * ddafa is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * ddafa is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with ddafa. If not, see <http://www.gnu.org/licenses/>.
 *
 * Date: 18 August 2016
 * Authors: Jan Stephan
 */

#include <cmath>
#include <cstddef>
#include <cstdint>
#include <functional>
#include <utility>

#include <boost/log/trivial.hpp>

#include <ddrf/cuda/algorithm.h>
#include <ddrf/cuda/coordinates.h>
#include <ddrf/cuda/launch.h>
#include <ddrf/cuda/memory.h>
#include <ddrf/cuda/sync_policy.h>
#include <ddrf/cuda/utility.h>

#include "exception.h"
#include "geometry.h"
#include "projection.h"
#include "reconstruction_stage.h"
#include "region_of_interest.h"
#include "scheduler.h"
#include "volume.h"

namespace ddafa
{
    namespace
    {
        // constants for the current subvolume -- these never change between kernel executions
        struct reconstruction_constants
        {
            std::uint32_t vol_dim_x;
            std::uint32_t vol_dim_x_full;
            std::uint32_t vol_dim_y;
            std::uint32_t vol_dim_y_full;
            std::uint32_t vol_dim_z;
            std::uint32_t vol_dim_z_full;
            std::uint32_t vol_offset;

            float l_vx_x;
            float l_vx_y;
            float l_vx_z;

            std::uint32_t proj_dim_x;
            std::uint32_t proj_dim_y;

            float l_px_x;
            float l_px_y;
            
            float delta_s;
            float delta_t;

            float d_so;
            float d_sd;
        };

        // note that each device will automatically keep track of its own symbol, no further synchronization needed
        __device__ __constant__ reconstruction_constants dev_consts__{};
        __device__ __constant__ region_of_interest dev_roi__{};

        inline __device__ auto vol_centered_coordinate(unsigned int coord, std::uint32_t dim, float size) -> float
        {
            auto size2 = size / 2.f;
            return -(dim * size2) + size2 + coord * size;
        }

        inline __device__ auto proj_real_coordinate(float coord, std::uint32_t dim, float size, float offset) -> float
        {
            auto size2 = size / 2.f;
            auto min = -(dim * size2) - offset;
            return (coord - min) / size - (1.f / 2.f);
        }

        template <bool enable_roi>
        __global__ void backproject(float* __restrict__ vol, std::size_t vol_pitch, hipTextureObject_t proj, float angle_sin, float angle_cos)
        {
            auto k = ddrf::cuda::coord_x();
            auto l = ddrf::cuda::coord_y();
            auto m = ddrf::cuda::coord_z();

            if((k < dev_consts__.vol_dim_x) && (l < dev_consts__.vol_dim_y) && (m < dev_consts__.vol_dim_z))
            {
                auto slice_pitch = vol_pitch * dev_consts__.vol_dim_y;
                auto slice = reinterpret_cast<char*>(vol) + m * slice_pitch;
                auto row = reinterpret_cast<float*>(slice + l * vol_pitch);

                // optimization hackery: load value from global memory while executing other instructions
                auto old_val = row[k];
                
                // add ROI offset. If enable_roi == false, the compiler will optimize this code away
                if(enable_roi)
                {
                    k += dev_roi__.x1;
                    l += dev_roi__.y1;
                    m += dev_roi__.z1;
                }

                // add offset for the current subvolume
                m += dev_consts__.vol_offset;

                // get centered coordinates -- volume center is at (0, 0, 0) and the top slice is at -(vol_d_off / 2)
                auto x_k = vol_centered_coordinate(k, dev_consts__.vol_dim_x_full, dev_consts__.l_vx_x);
                auto y_l = vol_centered_coordinate(l, dev_consts__.vol_dim_y_full, dev_consts__.l_vx_y);
                auto z_m = vol_centered_coordinate(m, dev_consts__.vol_dim_z_full, dev_consts__.l_vx_z);

                // rotate coordinates
                auto s = x_k * angle_cos + y_l * angle_sin;
                auto t = -x_k * angle_sin + y_l * angle_cos;

                // project rotated coordinates
                auto factor = dev_consts__.d_sd / (s + dev_consts__.d_so);
                // add 0.5 to each coordinate to deal with CUDA's filtering mechanism
                auto h = proj_real_coordinate(t * factor, dev_consts__.proj_dim_x, dev_consts__.l_px_x, dev_consts__.delta_s) + 0.5f;
                auto v = proj_real_coordinate(z_m * factor, dev_consts__.proj_dim_y, dev_consts__.l_px_y, dev_consts__.delta_t) + 0.5f;

                // get projection value (note the implicit linear interpolation)
                auto det = tex2D<float>(proj, h, v);

                // backproject
                auto u = -(dev_consts__.d_so / (s + dev_consts__.d_so));

                // restore old coordinate for writing. If enable_roi == false, the compiler will optimize this code away
                if(enable_roi)
                    k -= dev_roi__.x1;

                // write value
                row[k] = old_val + 0.5f * det * u * u;
            }
        }

        auto download(const ddrf::cuda::pitched_device_ptr<float>& in, ddrf::cuda::pinned_host_ptr<float>& out,
                        std::uint32_t x, std::uint32_t y, std::uint32_t z) -> void
        {
            ddrf::cuda::copy(ddrf::cuda::sync, out, in, x, y, z);
        }
    }

    reconstruction_stage::reconstruction_stage(int device) noexcept
    : device_{device}
    {
    }

    auto reconstruction_stage::assign_task(task t) noexcept -> void
    {
        det_geo_ = t.det_geo;
        vol_geo_ = t.vol_geo;
        subvol_geo_ = t.subvol_geo;
        enable_angles_ = t.enable_angles;

        enable_roi_ = t.enable_roi;
        roi_ = t.roi;

        task_id_ = t.id;
    }

    auto reconstruction_stage::run() -> void
    {
        auto sre = stage_runtime_error{"reconstruction_stage::run() failed"};

        try
        {
            ddrf::cuda::set_device(device_);

            auto dim_z = std::uint32_t{};
            // if this is the lowest subvolume we need to consider the remaining slices
            if(task_id_ == task_num_ - 1)
                dim_z = subvol_geo_.dim_z + subvol_geo_.remainder;
            else
                dim_z = subvol_geo_.dim_z;

            // create host volume
            auto vol_h_ptr = ddrf::cuda::make_unique_pinned_host<float>(subvol_geo_.dim_x, subvol_geo_.dim_y, dim_z);
            ddrf::cuda::fill(ddrf::cuda::sync, vol_h_ptr, 0, subvol_geo_.dim_x, subvol_geo_.dim_y, dim_z);

            // create device volume
            auto vol_d_ptr = ddrf::cuda::make_unique_device<float>(subvol_geo_.dim_x, subvol_geo_.dim_y, dim_z);
            ddrf::cuda::fill(ddrf::cuda::sync, vol_d_ptr, 0, subvol_geo_.dim_x, subvol_geo_.dim_y, dim_z);

            // calculate offset for the current subvolume
            auto offset = task_id_ * subvol_geo_.dim_z;

            // utility variables
            auto delta_s = det_geo_.delta_s * det_geo_.l_px_row;
            auto delta_t = det_geo_.delta_t * det_geo_.l_px_col;

            // initialize dev_consts__
            auto host_consts = reconstruction_constants {
                subvol_geo_.dim_x,
                vol_geo_.dim_x,
                subvol_geo_.dim_y,
                vol_geo_.dim_y,
                subvol_geo_.dim_z,
                vol_geo_.dim_z,
                offset,
                vol_geo_.l_vx_x,
                vol_geo_.l_vx_y,
                vol_geo_.l_vx_z,
                det_geo_.n_row,
                det_geo_.n_col,
                det_geo_.l_px_row,
                det_geo_.l_px_col,
                delta_s,
                delta_t,
                det_geo_.d_so,
                std::abs(det_geo_.d_so) + std::abs(det_geo_.d_od)
            };
            
            auto err = hipMemcpyToSymbol(HIP_SYMBOL(dev_consts__), &host_consts, sizeof(host_consts));
            if(err != hipSuccess)
            {
                BOOST_LOG_TRIVIAL(fatal) << "Could not initialize device constants: " << hipGetErrorString(err);
                throw stage_runtime_error{"reconstruction_stage::run() failed"};
            }

            // initialize dev_roi__
            if(enable_roi_)
            {
                err = hipMemcpyToSymbol(HIP_SYMBOL(dev_roi__), &roi_, sizeof(roi_));
                if(err != hipSuccess)
                {
                    BOOST_LOG_TRIVIAL(fatal) << "Could not initialize region of interest on device: " << hipGetErrorString(err);
                    throw stage_runtime_error{"reconstruction_stage::process() failed"};
                }
            }

            while(true)
            {
                auto p = input_();
                if(!p.valid)
                    break;

                if(p.idx % 10 == 0)
                    BOOST_LOG_TRIVIAL(info) << "Reconstruction processing projection #" << p.idx << " on device #" << device_ << " in stream " << p.stream;

                // get angular position of the current projection
                auto phi = 0.f;
                if(enable_angles_)
                    phi = p.phi;
                else
                    phi = static_cast<float>(p.idx) * det_geo_.delta_phi;

                // transform to radians
                phi *= static_cast<float>(M_PI) / 180.f;

                auto sin = std::sin(phi);
                auto cos = std::cos(phi);

                // create a CUDA texture from the projection
                auto res_desc = hipResourceDesc{};
                res_desc.resType = hipResourceTypePitch2D;
                res_desc.res.pitch2D.desc = hipCreateChannelDesc<float>();
                res_desc.res.pitch2D.devPtr = reinterpret_cast<void*>(p.ptr.get());
                res_desc.res.pitch2D.width = p.width;
                res_desc.res.pitch2D.height = p.height;
                res_desc.res.pitch2D.pitchInBytes = p.ptr.pitch();

                auto tex_desc = hipTextureDesc{};
                tex_desc.addressMode[0] = hipAddressModeBorder;
                tex_desc.addressMode[1] = hipAddressModeBorder;
                tex_desc.filterMode = hipFilterModeLinear;
                tex_desc.readMode = hipReadModeElementType;
                tex_desc.normalizedCoords = 0;

                auto tex = hipTextureObject_t{0};
                err = hipCreateTextureObject(&tex, &res_desc, &tex_desc, nullptr);
                if(err != hipSuccess)
                {
                    BOOST_LOG_TRIVIAL(fatal) << "Could not create CUDA texture: " << hipGetErrorString(err);
                    throw stage_runtime_error{"reconstruction_stage::process() failed"};
                }

                if(enable_roi_)
                    ddrf::cuda::launch_async(p.stream, subvol_geo_.dim_x, subvol_geo_.dim_y, subvol_geo_.dim_z,
                                        backproject<true>, vol_d_ptr.get(), vol_d_ptr.pitch(), tex, sin, cos);
                else
                    ddrf::cuda::launch_async(p.stream, subvol_geo_.dim_x, subvol_geo_.dim_y, subvol_geo_.dim_z,
                                        backproject<false>, vol_d_ptr.get(), vol_d_ptr.pitch(), tex, sin, cos);

                ddrf::cuda::synchronize_stream(p.stream);

                err = hipDestroyTextureObject(tex);
                if(err != hipSuccess)
                {
                    BOOST_LOG_TRIVIAL(fatal) << "Could not destroy CUDA texture: " << hipGetErrorString(err);
                    throw stage_runtime_error{"reconstruction_stage::process() failed"};
                }
            }

            // copy results to host
            download(vol_d_ptr, vol_h_ptr, subvol_geo_.dim_x, subvol_geo_.dim_y, subvol_geo_.dim_z);

            // create and move output volume -- done
            output_(output_type{std::move(vol_h_ptr), subvol_geo_.dim_x, subvol_geo_.dim_y, dim_z, offset , true, device_});
            BOOST_LOG_TRIVIAL(info) << "Completed task #" << task_id_ << " on device #" << device_;
        }
        catch(const ddrf::cuda::bad_alloc& ba)
        {
            BOOST_LOG_TRIVIAL(fatal) << "reconstruction_stage::run() encountered a bad_alloc: " << ba.what();
            throw sre;
        }
        catch(const ddrf::cuda::invalid_argument& ia)
        {
            BOOST_LOG_TRIVIAL(fatal) << "reconstruction_stage::run() passed an invalid argument to the CUDA runtime: " << ia.what();
            throw sre;
        }
        catch(const ddrf::cuda::runtime_error& re)
        {
            BOOST_LOG_TRIVIAL(fatal) << "reconstruction-stage::run() encountered a CUDA runtime error: " << re.what();
            throw sre;
        }
    }

    auto reconstruction_stage::set_input_function(std::function<input_type(void)> input) noexcept -> void
    {
        input_ = input;
    }

    auto reconstruction_stage::set_output_function(std::function<void(output_type)> output) noexcept -> void
    {
        output_ = output;
    }
}
