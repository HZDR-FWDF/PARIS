#include "hip/hip_runtime.h"
/*
 * This file is part of the ddafa reconstruction program.
 *
 * Copyright (C) 2016 Helmholtz-Zentrum Dresden-Rossendorf
 *
 * Licensed under the EUPL, Version 1.1 or - as soon they will be approved by
 * the European Commission - subsequent version of the EUPL (the "Licence");
 * You may not use this work except in compliance with the Licence.
 * You may obtain a copy of the Licence at:
 *
 * http://ec.europa.eu/idabc/eupl
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the Licence is distributed on an "AS IS" basis,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the Licence for the specific language governing permissions and
 * limitations under the Licence.
 *
 * Date: 18 August 2016
 * Authors: Jan Stephan
 */

#include <atomic>
#include <cmath>
#include <functional>
#include <future>
#include <thread>
#include <utility>
#include <vector>

#include <boost/log/trivial.hpp>

#include <ddrf/cuda/algorithm.h>
#include <ddrf/cuda/coordinates.h>
#include <ddrf/cuda/launch.h>
#include <ddrf/cuda/memory.h>
#include <ddrf/cuda/sync_policy.h>

#include "exception.h"
#include "geometry.h"
#include "metadata.h"
#include "reconstruction_stage.h"

namespace ddafa
{
    namespace
    {
        inline __device__ auto vol_centered_coordinate(unsigned int coord, std::size_t dim, float size) -> float
        {
            auto size2 = size / 2.f;
            return -(dim * size2) + size2 + coord * size;
        }

        // round and cast as needed
        inline __device__ auto proj_real_coordinate(float coord, std::size_t dim, float size, float offset) -> float
        {
            auto size2 = size / 2.f;
            auto min = -(dim * size2) - offset;
            return (coord - min) / size - (1.f / 2.f);
        }

        template <class T>
        inline __device__ auto as_unsigned(T x) -> unsigned int
        {
            return static_cast<unsigned int>(x);
        }

        __device__ auto interpolate(float h, float v, const float* proj, std::size_t proj_width, std::size_t proj_height, std::size_t proj_pitch,
                                    float pixel_size_x, float pixel_size_y, float offset_x, float offset_y)
        -> float
        {
            auto h_real = proj_real_coordinate(h, proj_width, pixel_size_x, offset_x);
            auto v_real = proj_real_coordinate(v, proj_height, pixel_size_y, offset_y);

            auto h_j0 = floorf(h_real);
            auto h_j1 = h_j0 + 1.f;
            auto v_i0 = floorf(v_real);
            auto v_i1 = v_i0 + 1.f;

            auto w_h0 = h_real - h_j0;
            auto w_v0 = v_real - v_i0;

            auto w_h1 = 1.f - w_h0;
            auto w_v1 = 1.f - w_v0;

            auto h_j0_ui = as_unsigned(h_j0);
            auto h_j1_ui = as_unsigned(h_j1);
            auto v_i0_ui = as_unsigned(v_i0);
            auto v_i1_ui = as_unsigned(v_i1);

            // ui coordinates might be invalid due to negative v_i0, thus
            // bounds checking
            auto h_j0_valid = (h_j0 >= 0.f);
            auto h_j1_valid = (h_j1 < static_cast<float>(proj_width));
            auto v_i0_valid = (v_i0 >= 0.f);
            auto v_i1_valid = (v_i1 < static_cast<float>(proj_height));

            auto upper_row = reinterpret_cast<const float*>(reinterpret_cast<const char*>(proj) + v_i0_ui * proj_pitch);
            auto lower_row = reinterpret_cast<const float*>(reinterpret_cast<const char*>(proj) + v_i1_ui * proj_pitch);

            auto tl = 0.f;
            auto bl = 0.f;
            auto tr = 0.f;
            auto br = 0.f;
            if(h_j0_valid && h_j1_valid && v_i0_valid && v_i1_valid)
            {
                tl = upper_row[h_j0_ui];
                bl = lower_row[h_j0_ui];
                tr = upper_row[h_j1_ui];
                br = lower_row[h_j1_ui];
            }

            auto val =  w_h1    * w_v1  * tl +
                        w_h1    * w_v0  * bl +
                        w_h0    * w_v1  * tr +
                        w_h0    * w_v0  * br;

            return val;
        }

        __global__ void backproject(float* __restrict__ vol, std::size_t vol_w, std::size_t vol_h, std::size_t vol_d, std::size_t vol_pitch,
                                    std::size_t vol_offset, std::size_t vol_d_full, float voxel_size_x, float voxel_size_y, float voxel_size_z,
                                    const float* __restrict__ proj, std::size_t proj_w, std::size_t proj_h, std::size_t proj_pitch,
                                    float pixel_size_x, float pixel_size_y, float pixel_offset_x, float pixel_offset_y,
                                    float angle_sin, float angle_cos, float dist_src, float dist_sd)
        {
            auto k = ddrf::cuda::coord_x();
            auto l = ddrf::cuda::coord_y();
            auto m = ddrf::cuda::coord_z();

            if((k < vol_w) && (l < vol_h) && (m < vol_d))
            {
                auto slice_pitch = vol_pitch * vol_h;
                auto slice = reinterpret_cast<char*>(vol) + m * slice_pitch;
                auto row = reinterpret_cast<float*>(slice + l * vol_pitch);

                // add offset for the current subvolume
                auto m_off = m + vol_offset;

                // get centered coordinates -- volume center is at (0, 0, 0) and the top slice is at -(vol_d_off / 2)
                auto x_k = vol_centered_coordinate(k, vol_w, voxel_size_x);
                auto y_l = vol_centered_coordinate(l, vol_h, voxel_size_y);
                auto z_m = vol_centered_coordinate(m_off, vol_d_full, voxel_size_z);

                // rotate coordinates
                auto s = x_k * angle_cos + y_l * angle_sin;
                auto t = -x_k * angle_sin + y_l * angle_cos;
                auto z = z_m;

                // project rotated coordinates
                auto factor = dist_sd / (s + dist_src);
                auto h = t * factor;
                auto v = z * factor;

                // get projection value by interpolation
                auto det = interpolate(h, v, proj, proj_w, proj_h, proj_pitch, pixel_size_x, pixel_size_y, pixel_offset_x, pixel_offset_y);

                // backproject
                auto u = -(dist_src / (s + dist_src));
                row[k] += 0.5f * det * powf(u, 2.f);
            }
        }
    }

    reconstruction_stage::reconstruction_stage(const geometry& det_geo, const volume_metadata& vol_geo, const std::vector<volume_metadata>& subvol_geos,
                                                bool predefined_angles)
    : det_geo_(det_geo), vol_geo_(vol_geo), predefined_angles_{predefined_angles}
    {
        auto err = hipGetDeviceCount(&devices_);
        if(err != hipSuccess)
        {
            BOOST_LOG_TRIVIAL(fatal) << "reconstruction_stage::reconstruction_stage() could not obtain devices: " << hipGetErrorString(err);
            throw stage_construction_error{"reconstruction_stage::reconstruction_stage() failed"};
        }

        try
        {
            using sv_size_type = typename decltype(subvol_vec_)::size_type;
            auto d_sv = static_cast<sv_size_type>(devices_);
            subvol_vec_ = decltype(subvol_vec_){d_sv};

            using svg_size_type = typename decltype(subvol_geo_vec_)::size_type;
            auto d_svg = static_cast<svg_size_type>(devices_);
            subvol_geo_vec_ = decltype(subvol_geo_vec_){d_svg};

            using iv_size_type = typename decltype(input_vec_)::size_type;
            auto d_iv = static_cast<iv_size_type>(devices_);
            input_vec_ = decltype(input_vec_){d_iv};

            vol_out_.first = ddrf::cuda::make_unique_pinned_host<float>(vol_geo_.width, vol_geo_.height, vol_geo_.depth);
            ddrf::cuda::fill(ddrf::cuda::async, vol_out_.first, 0, vol_geo_.width, vol_geo_.height, vol_geo_.depth);

            vol_out_.second = vol_geo;
            vol_out_.second.valid = true;

            for(auto i = 0; i < devices_; ++i)
            {
                err = hipSetDevice(i);
                if(err != hipSuccess)
                {
                    BOOST_LOG_TRIVIAL(fatal) << "reconstruction_stage::reconstruction_stage could not set CUDA device: " << hipGetErrorString(err);
                    throw stage_construction_error{"reconstruction_stage::reconstruction_stage() failed"};
                }

                for(const auto& g : subvol_geos)
                {
                    if(g.device == i)
                    {
                        auto ptr = ddrf::cuda::make_unique_device<float>(g.width, g.height, g.depth + g.remainder);
                        ddrf::cuda::fill(ddrf::cuda::async, ptr, 0, g.width, g.height, g.depth + g.remainder);

                        d_sv = static_cast<sv_size_type>(g.device);
                        subvol_vec_[d_sv] = std::make_pair(std::move(ptr), g);

                        d_svg = static_cast<svg_size_type>(g.device);
                        subvol_geo_vec_[d_svg] = g;
                        break;
                    }
                }
            }
        }
        catch(const ddrf::cuda::bad_alloc& ba)
        {
            BOOST_LOG_TRIVIAL(fatal) << "reconstruction_stage::reconstruction_stage() could not allocate memory: " << ba.what();
            throw stage_construction_error{"reconstruction_stage::reconstruction_stage() failed"};
        }
    }

    reconstruction_stage::reconstruction_stage(reconstruction_stage&& other) noexcept
    : input_{std::move(other.input_)}, output_{std::move(other.output_)}
    , det_geo_(other.det_geo_), vol_geo_(other.vol_geo_), predefined_angles_(other.predefined_angles_), vol_out_{std::move(other.vol_out_)}
    , devices_{other.devices_}, subvol_vec_{std::move(other.subvol_vec_)}, subvol_geo_vec_{std::move(other.subvol_geo_vec_)}, input_vec_{std::move(other.input_vec_)}
    {
        if(other.lock_.test_and_set())
            lock_.test_and_set();
        else
            lock_.clear();
    }

    reconstruction_stage::~reconstruction_stage()
    {
        for(auto&& s : subvol_vec_)
        {
            hipSetDevice(s.second.device);
            s.first.reset(nullptr);
        }
    }

    auto reconstruction_stage::operator=(reconstruction_stage&& other) noexcept -> reconstruction_stage&
    {
        input_ = std::move(other.input_);
        output_ = std::move(other.output_);
        det_geo_ = other.det_geo_;
        vol_geo_ = other.vol_geo_;
        predefined_angles_ = other.predefined_angles_;
        vol_out_ = std::move(other.vol_out_);
        devices_ = other.devices_;
        subvol_vec_ = std::move(other.subvol_vec_);
        subvol_geo_vec_ = std::move(other.subvol_geo_vec_);
        input_vec_ = std::move(other.input_vec_);

        if(other.lock_.test_and_set())
            lock_.test_and_set();
        else
            lock_.clear();

        return *this;
    }

    auto reconstruction_stage::run() -> void
    {
        try
        {
            std::vector<std::future<void>> futures;
            for(int i = 0; i < devices_; ++i)
                futures.emplace_back(std::async(std::launch::async, &reconstruction_stage::process, this, i));

            while(true)
            {
                auto proj = input_();
                auto valid = proj.second.valid;
                safe_push(std::move(proj));
                if(!valid)
                    break;
            }

            for(auto&& f: futures)
                f.get();

            output_(std::move(vol_out_));
            BOOST_LOG_TRIVIAL(info) << "Reconstruction complete.";
        }
        catch(const stage_runtime_error& sre)
        {
            BOOST_LOG_TRIVIAL(fatal) << "reconstruction_stage::run() failed to execute: " << sre.what();
            throw stage_runtime_error{"reconstruction_stage::run() failed"};
        }
    }

    auto reconstruction_stage::set_input_function(std::function<input_type(void)> input) noexcept -> void
    {
        input_ = input;
    }

    auto reconstruction_stage::set_output_function(std::function<void(output_type)> output) noexcept -> void
    {
        output_ = output;
    }

    auto reconstruction_stage::safe_push(input_type proj) -> void
    {
        while(lock_.test_and_set(std::memory_order_acquire))
            std::this_thread::yield();

        if(proj.second.valid)
            input_vec_[proj.second.device].push(std::move(proj));
        else
        {
            for(auto i = 0; i < devices_; ++i)
                input_vec_[i].push(input_type{});
        }

        lock_.clear(std::memory_order_release);
    }

    auto reconstruction_stage::safe_pop(int device) -> input_type
    {
        while(input_vec_.empty())
            std::this_thread::yield();

        auto& queue = input_vec_[device];
        while(queue.empty())
            std::this_thread::yield();

        while(lock_.test_and_set(std::memory_order_acquire))
            std::this_thread::yield();

        auto proj = std::move(queue.front());
        queue.pop();

        lock_.clear(std::memory_order_release);
        return proj;
    }

    auto reconstruction_stage::process(int device) -> void
    {
        auto err = hipSetDevice(device);
        if(err != hipSuccess)
        {
            BOOST_LOG_TRIVIAL(fatal) << "reconstruction_stage::process() could not set CUDA device: " << hipGetErrorString(err);
            throw stage_runtime_error{"reconstruction_stage::process() failed"};
        }

        try
        {
            auto vol_count = typename decltype(subvol_geo_vec_)::size_type{0};
            auto first = true;

            auto delta_s = det_geo_.delta_s * det_geo_.l_px_row;
            auto delta_t = det_geo_.delta_t * det_geo_.l_px_col;
            while(true)
            {
                using svg_size_type = typename decltype(subvol_geo_vec_)::size_type;
                auto d_svg = static_cast<svg_size_type>(device);
                auto v_geo = subvol_geo_vec_.at(d_svg);

                using sv_size_type = typename decltype(subvol_vec_)::size_type;
                auto d_sv = static_cast<sv_size_type>(device);
                auto& v = subvol_vec_[d_sv];

                auto p = safe_pop(device);
                if(!p.second.valid)
                {
                    download_and_reset(device, vol_count);
                    break;
                }

                if(p.second.index == 0)
                {
                    if(first)
                        first = false;
                    else
                    {
                        download_and_reset(device, vol_count);
                        ++vol_count;
                    }
                }

                if(p.second.index % 10 == 0)
                    BOOST_LOG_TRIVIAL(info) << "Reconstruction processing projection #" << p.second.index << " on device #" << device;

                auto phi = 0.f;
                if(predefined_angles_)
                    phi = p.second.phi;
                else
                    phi = p.second.index * det_geo_.delta_phi;
                auto phi_rad = phi * M_PI / 180.f;
                auto sin = static_cast<float>(std::sin(phi_rad));
                auto cos = static_cast<float>(std::cos(phi_rad));

                auto offset = v_geo.offset * vol_count;

                auto v_ptr = v.first.get();
                auto p_ptr = static_cast<const float*>(p.first.get());
                ddrf::cuda::launch(v.second.width, v.second.height, v.second.depth,
                                    backproject,
                                    v_ptr, v.second.width, v.second.height, v.second.depth, v.first.pitch(), offset, vol_geo_.depth,
                                        v.second.vx_size_x, v.second.vx_size_y, v.second.vx_size_z,
                                    p_ptr, p.second.width, p.second.height, p.first.pitch(), det_geo_.l_px_row, det_geo_.l_px_col,
                                        delta_s, delta_t,
                                    sin, cos, det_geo_.d_so, std::abs(det_geo_.d_so) + std::abs(det_geo_.d_od));
            }
        }
        catch(const ddrf::cuda::bad_alloc& ba)
        {
            BOOST_LOG_TRIVIAL(fatal) << "reconstruction_stage::process() encountered a bad_alloc: " << ba.what();
            throw stage_runtime_error{"reconstruction_stage::process() failed"};
        }
        catch(const ddrf::cuda::invalid_argument ia)
        {
            BOOST_LOG_TRIVIAL(fatal) << "reconstruction_stage::process() passed an invalid argument to the CUDA runtime: " << ia.what();
            throw stage_runtime_error{"reconstruction_stage::process() failed"};
        }
        catch(const ddrf::cuda::runtime_error& re)
        {
            BOOST_LOG_TRIVIAL(fatal) << "reconstruction-stage::process() encountered a CUDA runtime error: " << re.what();
            throw stage_runtime_error{"reconstruction_stage::process() failed"};
        }
    }

    auto reconstruction_stage::download_and_reset(int device, std::uint32_t vol_count) -> void
    {
        using sv_size_type = typename decltype(subvol_vec_)::size_type;
        auto d_sv = static_cast<sv_size_type>(device);
        auto& v = subvol_vec_[d_sv];

        try
        {
            ddrf::cuda::copy(ddrf::cuda::sync, vol_out_.first, v.first, v.second.width, v.second.height, v.second.depth + v.second.remainder,
                                0, 0, vol_count * v.second.offset);

            BOOST_LOG_TRIVIAL(debug) << "Copy succeeded";

            ddrf::cuda::fill(ddrf::cuda::sync, v.first, 0, v.second.width, v.second.height, v.second.depth);

            BOOST_LOG_TRIVIAL(debug) << "Memset succeeded";
        }
        catch(const ddrf::cuda::invalid_argument& ia)
        {
            BOOST_LOG_TRIVIAL(fatal) << "reconstruction_stage::download_and_reset() passed an invalid argument to the CUDA runtime: " << ia.what();
            throw stage_runtime_error{"reconstruction_stage::download_and_reset() failed"};
        }
    }
}
