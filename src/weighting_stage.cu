#include "hip/hip_runtime.h"
/*
 * This file is part of the ddafa reconstruction program.
 *
 * Copyright (C) 2016 Helmholtz-Zentrum Dresden-Rossendorf
 *
 * Licensed under the EUPL, Version 1.1 or - as soon they will be approved by
 * the European Commission - subsequent version of the EUPL (the "Licence");
 * You may not use this work except in compliance with the Licence.
 * You may obtain a copy of the Licence at:
 *
 * http://ec.europa.eu/idabc/eupl
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the Licence is distributed on an "AS IS" basis,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the Licence for the specific language governing permissions and
 * limitations under the Licence.
 *
 * Date: 18 August 2016
 * Authors: Jan Stephan
 */

#include <atomic>
#include <cmath>
#include <functional>
#include <future>
#include <map>
#include <thread>
#include <utility>

#include <boost/log/trivial.hpp>

#include <ddrf/cuda/coordinates.h>
#include <ddrf/cuda/launch.h>

#include "exception.h"
#include "metadata.h"
#include "weighting_stage.h"

namespace ddafa
{
    namespace kernel
    {
        __global__ void weight(float* output, const float* input,
                                std::size_t n_row, std::size_t n_col, std::size_t pitch,
                                float h_min, float v_min,
                                float d_sd,
                                float l_px_row, float l_px_col)
        {
            auto s = ddrf::cuda::coord_x();
            auto t = ddrf::cuda::coord_y();

            if((s < n_row) && (t < n_col))
            {
                auto input_row = reinterpret_cast<const float*>(reinterpret_cast<const char*>(input) + t * pitch);
                auto output_row = reinterpret_cast<float*>(reinterpret_cast<char*>(input) + t * pitch);

                // detector coordinates in mm
                auto h_s = (l_px_row / 2) + s * l_px_row + h_min;
                auto v_t = (l_px_col / 2) + t * l_px_col + v_min;

                // calculate weight
                auto w_st = d_sd * rsqrtf(powf(d_sd, 2) + powf(h_s, 2) + powf(v_t, 2));

                // write value
                output_row[s] = input_row[s] * w_st;
            }
        }
    }

    weighting_stage::weighting_stage(std::uint32_t n_row, std::uint32_t n_col,
                                        float l_px_row, float l_px_col,
                                        float delta_s, float delta_t,
                                        float d_so, float d_od) noexcept
    {
        h_min_ = delta_s * l_px_row - n_row * l_px_row / 2;
        v_min_ = delta_t * l_px_col - n_col * l_px_col / 2;
        d_sd_ = std::abs(d_so) + std::abs(d_od);
    }

    auto weighting_stage::run() -> void
    {
        auto devices = int{};
        auto err = hipGetDeviceCount(&devices);
        if(err != hipSuccess)
        {
            BOOST_LOG_TRIVIAL(fatal) << "weighting_stage::run() could not obtain CUDA devices: " << hipGetErrorString(err);
            throw stage_runtime_error{"weighting_stage::run() failed to initialize"};
        }

        std::map<int, std::future<void>> futures;
        for(int i = 0; i < devices; ++i)
            futures[i] = std::async(std::launch::async, &weighting_stage::process, this, i);

        while(true)
        {

            auto proj = input_();
            while(lock_.test_and_set(std::memory_order_acquire))
                std::this_thread::yield();

            if(proj.second.valid)
                input_map_[proj.second.device].push(std::move(proj));
            else
            {
                for(auto i = 0; i < devices; ++i)
                    input_map_[i].push(input_type());
            }

            lock_.clear(std::memory_order_release);
        }

        try
        {
            for(auto&& fp : futures)
                fp.second.get();
        }
        catch(const stage_runtime_error& sre)
        {
            BOOST_LOG_TRIVIAL(fatal) << "weighting_stage::run() failed to execute: " << sre.what();
            throw stage_runtime_error{"weighting_stage::run() failed"};
        }

        output_(std::make_pair(nullptr, projection_metadata{0, 0, 0, 0.f, false, 0}));
        BOOST_LOG_TRIVIAL(info) << "Weighted all projections.";
    }

    auto weighting_stage::set_input_function(std::function<input_type(void)> input) noexcept -> void
    {
        input_ = input;
    }

    auto weighting_stage::set_output_function(std::function<void(output_type)> output) noexcept -> void
    {
        output_ = output;
    }

    auto weighting_stage::process(int device) -> void
    {
        auto err = hipSetDevice(device);
        if(err != hipSuccess)
        {
            BOOST_LOG_TRIVIAL(fatal) << "weighting_stage::process() could not set device: " << hipGetErrorString(err);
            throw stage_runtime_error{"weighting_stage::process() failed to initialize"};
        }

        while(true)
        {
            while(input_map_.count(device) == 0)
                std::this_thread::yield();

            while(lock_.test_and_set(std::memory_order_acquire))
                std::this_thread::yield();

            auto& queue = input_map_.at(device);
            if(queue.empty())
            {
                lock_.clear(std::memory_order_release);
                continue;
            }
            auto proj = std::move(queue.front());
            queue.pop();

            lock_.clear(std::memory_order_release);

            if(!proj.second.valid)
                break;

            try
            {
                ddrf::cuda::launch(proj.second.width, proj.second.height,
                                    kernel::weight,
                                    proj.first.get(), static_cast<const float*>(proj.first.get()),
                                    n_row_, n_col_, proj.first.pitch(),
                                    h_min_, v_min_,
                                    d_sd_,
                                    l_px_row_, l_px_col_);
            }
            catch(const ddrf::cuda::bad_alloc& ba)
            {
                BOOST_LOG_TRIVIAL(fatal) << "weighting_stage::process() encountered bad_alloc while invoking kernel: " << ba.what();
                throw stage_runtime_error{"weighting_stage::process(): weighting kernel failed"};
            }
            catch(const ddrf::cuda::runtime_error& re)
            {
                BOOST_LOG_TRIVIAL(fatal) << "weighting_stage::process() encountered runtime_error while invoking kernel: " << re.what();
                throw stage_runtime_error{"weighting_stage::process(): weighting kernel failed"};
            }
            catch(const ddrf::cuda::invalid_argument& ia)
            {
                BOOST_LOG_TRIVIAL(fatal) << "weighting_stage::process() encountered invalid_argument while invoking kernel: " << ia.what();
                throw stage_runtime_error{"weighting_stage::process(): weighting kernel failed"};
            }

            output_(std::move(proj));
        }
    }
}
